#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>

#define C 64
#define N 64
#define H 56
#define W 56

#define R 3
#define S 3
using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[56];
  __shared__ float pad_temp_shared[960];
  __shared__ float kernel_shared[384];
  float pad_temp_shared_local[28];
  float kernel_shared_local[2];
  #pragma unroll
  for (int xx_c_init = 0; xx_c_init < 2; ++xx_c_init) {
    compute_local[(xx_c_init)] = 0.000000e+00f;
    compute_local[((xx_c_init + 28))] = 0.000000e+00f;
    compute_local[((xx_c_init + 2))] = 0.000000e+00f;
    compute_local[((xx_c_init + 30))] = 0.000000e+00f;
    compute_local[((xx_c_init + 4))] = 0.000000e+00f;
    compute_local[((xx_c_init + 32))] = 0.000000e+00f;
    compute_local[((xx_c_init + 6))] = 0.000000e+00f;
    compute_local[((xx_c_init + 34))] = 0.000000e+00f;
    compute_local[((xx_c_init + 8))] = 0.000000e+00f;
    compute_local[((xx_c_init + 36))] = 0.000000e+00f;
    compute_local[((xx_c_init + 10))] = 0.000000e+00f;
    compute_local[((xx_c_init + 38))] = 0.000000e+00f;
    compute_local[((xx_c_init + 12))] = 0.000000e+00f;
    compute_local[((xx_c_init + 40))] = 0.000000e+00f;
    compute_local[((xx_c_init + 14))] = 0.000000e+00f;
    compute_local[((xx_c_init + 42))] = 0.000000e+00f;
    compute_local[((xx_c_init + 16))] = 0.000000e+00f;
    compute_local[((xx_c_init + 44))] = 0.000000e+00f;
    compute_local[((xx_c_init + 18))] = 0.000000e+00f;
    compute_local[((xx_c_init + 46))] = 0.000000e+00f;
    compute_local[((xx_c_init + 20))] = 0.000000e+00f;
    compute_local[((xx_c_init + 48))] = 0.000000e+00f;
    compute_local[((xx_c_init + 22))] = 0.000000e+00f;
    compute_local[((xx_c_init + 50))] = 0.000000e+00f;
    compute_local[((xx_c_init + 24))] = 0.000000e+00f;
    compute_local[((xx_c_init + 52))] = 0.000000e+00f;
    compute_local[((xx_c_init + 26))] = 0.000000e+00f;
    compute_local[((xx_c_init + 54))] = 0.000000e+00f;
  }
  for (int rc_outer = 0; rc_outer < 4; ++rc_outer) {
    for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
      __syncthreads();
      #pragma unroll
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 120; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
        pad_temp_shared[((((((int)threadIdx.z) * 240) + (((int)threadIdx.y) * 120)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((1 <= (((((int)blockIdx.y) * 2) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner % 60) / 30)) + ry_outer)) && ((((((int)blockIdx.y) * 2) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner % 60) / 30)) + ry_outer) < 57)) && (1 <= ((((int)blockIdx.x) * 28) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner % 30)))) && (((((int)blockIdx.x) * 28) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner % 30)) < 57)) ? data[(((((((((((rc_outer * 50176) + (((int)threadIdx.z) * 12544)) + (((int)threadIdx.y) * 6272)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner / 60) * 3136)) + (((int)blockIdx.y) * 112)) + (((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner % 60) / 30) * 56)) + (ry_outer * 56)) + (((int)blockIdx.x) * 28)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner % 30)) - 57))] : 0.000000e+00f);
      }
      #pragma unroll
      for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 48; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
        kernel_shared[((((((int)threadIdx.z) * 96) + (((int)threadIdx.y) * 48)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = kernel[((((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 1152)) + (((int)threadIdx.y) * 576)) + (rc_outer * 144)) + ((ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 / 3) * 9)) + (ry_outer * 3)) + (ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 % 3)))];
      }
      __syncthreads();
      for (int rc_inner_outer = 0; rc_inner_outer < 16; ++rc_inner_outer) {
        #pragma unroll
        for (int rx_inner_outer = 0; rx_inner_outer < 3; ++rx_inner_outer) {
          #pragma unroll
          for (int ax3 = 0; ax3 < 2; ++ax3) {
            pad_temp_shared_local[(ax3)] = pad_temp_shared[(((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer))];
            pad_temp_shared_local[((ax3 + 2))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 2))];
            pad_temp_shared_local[((ax3 + 4))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 4))];
            pad_temp_shared_local[((ax3 + 6))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 6))];
            pad_temp_shared_local[((ax3 + 8))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 8))];
            pad_temp_shared_local[((ax3 + 10))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 10))];
            pad_temp_shared_local[((ax3 + 12))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 12))];
            pad_temp_shared_local[((ax3 + 14))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 14))];
            pad_temp_shared_local[((ax3 + 16))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 16))];
            pad_temp_shared_local[((ax3 + 18))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 18))];
            pad_temp_shared_local[((ax3 + 20))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 20))];
            pad_temp_shared_local[((ax3 + 22))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 22))];
            pad_temp_shared_local[((ax3 + 24))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 24))];
            pad_temp_shared_local[((ax3 + 26))] = pad_temp_shared[((((((rc_inner_outer * 60) + (((int)threadIdx.y) * 30)) + ax3) + rx_inner_outer) + 26))];
          }
          kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 48) + (rc_inner_outer * 3)) + rx_inner_outer))];
          kernel_shared_local[(1)] = kernel_shared[(((((((int)threadIdx.z) * 48) + (rc_inner_outer * 3)) + rx_inner_outer) + 192))];
          #pragma unroll
          for (int xx_c = 0; xx_c < 2; ++xx_c) {
            compute_local[(xx_c)] = (compute_local[(xx_c)] + (pad_temp_shared_local[(xx_c)] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 28))] = (compute_local[((xx_c + 28))] + (pad_temp_shared_local[(xx_c)] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 2))] = (compute_local[((xx_c + 2))] + (pad_temp_shared_local[((xx_c + 2))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 30))] = (compute_local[((xx_c + 30))] + (pad_temp_shared_local[((xx_c + 2))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 4))] = (compute_local[((xx_c + 4))] + (pad_temp_shared_local[((xx_c + 4))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 32))] = (compute_local[((xx_c + 32))] + (pad_temp_shared_local[((xx_c + 4))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 6))] = (compute_local[((xx_c + 6))] + (pad_temp_shared_local[((xx_c + 6))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 34))] = (compute_local[((xx_c + 34))] + (pad_temp_shared_local[((xx_c + 6))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 8))] = (compute_local[((xx_c + 8))] + (pad_temp_shared_local[((xx_c + 8))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 36))] = (compute_local[((xx_c + 36))] + (pad_temp_shared_local[((xx_c + 8))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 10))] = (compute_local[((xx_c + 10))] + (pad_temp_shared_local[((xx_c + 10))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 38))] = (compute_local[((xx_c + 38))] + (pad_temp_shared_local[((xx_c + 10))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 12))] = (compute_local[((xx_c + 12))] + (pad_temp_shared_local[((xx_c + 12))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 40))] = (compute_local[((xx_c + 40))] + (pad_temp_shared_local[((xx_c + 12))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 14))] = (compute_local[((xx_c + 14))] + (pad_temp_shared_local[((xx_c + 14))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 42))] = (compute_local[((xx_c + 42))] + (pad_temp_shared_local[((xx_c + 14))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 16))] = (compute_local[((xx_c + 16))] + (pad_temp_shared_local[((xx_c + 16))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 44))] = (compute_local[((xx_c + 44))] + (pad_temp_shared_local[((xx_c + 16))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 18))] = (compute_local[((xx_c + 18))] + (pad_temp_shared_local[((xx_c + 18))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 46))] = (compute_local[((xx_c + 46))] + (pad_temp_shared_local[((xx_c + 18))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 20))] = (compute_local[((xx_c + 20))] + (pad_temp_shared_local[((xx_c + 20))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 48))] = (compute_local[((xx_c + 48))] + (pad_temp_shared_local[((xx_c + 20))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 22))] = (compute_local[((xx_c + 22))] + (pad_temp_shared_local[((xx_c + 22))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 50))] = (compute_local[((xx_c + 50))] + (pad_temp_shared_local[((xx_c + 22))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 24))] = (compute_local[((xx_c + 24))] + (pad_temp_shared_local[((xx_c + 24))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 52))] = (compute_local[((xx_c + 52))] + (pad_temp_shared_local[((xx_c + 24))] * kernel_shared_local[(1)]));
            compute_local[((xx_c + 26))] = (compute_local[((xx_c + 26))] + (pad_temp_shared_local[((xx_c + 26))] * kernel_shared_local[(0)]));
            compute_local[((xx_c + 54))] = (compute_local[((xx_c + 54))] + (pad_temp_shared_local[((xx_c + 26))] * kernel_shared_local[(1)]));
          }
        }
      }
    }
  }
  #pragma unroll
  for (int xx_inner_inner_inner = 0; xx_inner_inner_inner < 2; ++xx_inner_inner_inner) {
    compute[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner))] = compute_local[(xx_inner_inner_inner)];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12544))] = compute_local[((xx_inner_inner_inner + 28))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 2))] = compute_local[((xx_inner_inner_inner + 2))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12546))] = compute_local[((xx_inner_inner_inner + 30))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 4))] = compute_local[((xx_inner_inner_inner + 4))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12548))] = compute_local[((xx_inner_inner_inner + 32))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 6))] = compute_local[((xx_inner_inner_inner + 6))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12550))] = compute_local[((xx_inner_inner_inner + 34))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 8))] = compute_local[((xx_inner_inner_inner + 8))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12552))] = compute_local[((xx_inner_inner_inner + 36))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 10))] = compute_local[((xx_inner_inner_inner + 10))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12554))] = compute_local[((xx_inner_inner_inner + 38))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12))] = compute_local[((xx_inner_inner_inner + 12))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12556))] = compute_local[((xx_inner_inner_inner + 40))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 14))] = compute_local[((xx_inner_inner_inner + 14))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12558))] = compute_local[((xx_inner_inner_inner + 42))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 16))] = compute_local[((xx_inner_inner_inner + 16))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12560))] = compute_local[((xx_inner_inner_inner + 44))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 18))] = compute_local[((xx_inner_inner_inner + 18))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12562))] = compute_local[((xx_inner_inner_inner + 46))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 20))] = compute_local[((xx_inner_inner_inner + 20))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12564))] = compute_local[((xx_inner_inner_inner + 48))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 22))] = compute_local[((xx_inner_inner_inner + 22))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12566))] = compute_local[((xx_inner_inner_inner + 50))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 24))] = compute_local[((xx_inner_inner_inner + 24))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12568))] = compute_local[((xx_inner_inner_inner + 52))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 26))] = compute_local[((xx_inner_inner_inner + 26))];
    compute[((((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)blockIdx.y) * 112)) + (((int)threadIdx.y) * 56)) + (((int)blockIdx.x) * 28)) + xx_inner_inner_inner) + 12570))] = compute_local[((xx_inner_inner_inner + 54))];
  }
}




float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
    #pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
void pad_input(float * x, float *y){
    #pragma omp parallel for
    for(unsigned int i=0;i<(H + 2)*(W+2)*C;++i){
        y[i] = 0.0f;
    }
    #pragma omp parallel for
    for(unsigned int c=0;c<C;++c){
        for(unsigned int h=0;h<H;++h){
            for(unsigned int w=0;w<W;++w){
                unsigned int h_padded = h + 1;
                unsigned int w_padded = w + 1;
                y[c*(H+2)*(W+2) + h_padded*(W+2) + w_padded] = x[c*(H)*(W) + h*(W) + w];
            }
        }
    }
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float * padded_input = new float[C*(H+2)*(W+2)];
    pad_input(input, padded_input);
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);


    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);

    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);

    dim3 grid(2,28,8);
    dim3 block(1,2,4);

    float * paddedInputDevice;
    chkerr(hipMalloc(&paddedInputDevice, C * (H + 2) * (W + 2) * sizeof(float)));
    chkerr(hipMemcpy(paddedInputDevice, padded_input, C * (H + 2) * (W + 2) * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<endl;
    return 0;
}


