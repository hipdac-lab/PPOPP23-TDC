#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>

#define C 128
#define N 96
#define H 14
#define W 14

#define R 3
#define S 3
using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[6];
  __shared__ float pad_temp_shared[224];
  __shared__ float kernel_shared[576];
  float pad_temp_shared_local[4];
  float kernel_shared_local[6];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[(((((int)threadIdx.z) * 28) + (((int)threadIdx.x) * 4)))] = ((((1 <= ((((int)blockIdx.y) * 2) + ((((int)threadIdx.x) * 4) / 7))) && (((((int)blockIdx.y) * 2) + ((((int)threadIdx.x) * 4) / 7)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + ((((int)threadIdx.x) * 4) % 7)))) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (((((int)threadIdx.x) * 4) / 7) * 14)) + (((int)blockIdx.x) * 7)) + ((((int)threadIdx.x) * 4) % 7)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 28) + (((int)threadIdx.x) * 4)) + 1))] = ((((1 <= ((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 4) + 1) / 7))) && (((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 4) + 1) / 7)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 1) % 7)))) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.x) * 4) + 1) / 7) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 1) % 7)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 28) + (((int)threadIdx.x) * 4)) + 2))] = ((((1 <= ((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 4) + 2) / 7))) && (((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 4) + 2) / 7)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 2) % 7)))) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.x) * 4) + 2) / 7) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 2) % 7)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 28) + (((int)threadIdx.x) * 4)) + 3))] = ((((1 <= ((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 4) + 3) / 7))) && (((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 4) + 3) / 7)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 3) % 7)))) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.x) * 4) + 3) / 7) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 3) % 7)) - 15))] : 0.000000e+00f);
    kernel_shared[(((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)))] = kernel[((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + (((((int)threadIdx.x) * 11) / 24) * 1152)) + (rc_outer * 72)) + (((((int)threadIdx.x) * 11) % 24) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 1))] = kernel[((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 1) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 1) % 24) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 2))] = kernel[((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 2) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 2) % 24) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 3))] = kernel[((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 3) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 3) % 24) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 4))] = kernel[((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 4) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 4) % 24) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 5))] = kernel[((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 5) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 5) % 24) * 3)))];
    if (((((int)threadIdx.z) * 3) + (((((int)threadIdx.x) * 11) + 6) / 24)) < 24) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 190) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 570) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 6))] = kernel[((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 6) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 6) % 24) * 3)))];
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 3) + (((((int)threadIdx.x) * 11) + 7) / 24)) < 24) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 7) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 569) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 7))] = kernel[((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 7) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 7) % 24) * 3)))];
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 3) + (((((int)threadIdx.x) * 11) + 8) / 24)) < 24) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 8) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 568) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 8))] = kernel[((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 8) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 8) % 24) * 3)))];
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 3) + (((((int)threadIdx.x) * 11) + 9) / 24)) < 24) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 189) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 567) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 9))] = kernel[((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 9) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 9) % 24) * 3)))];
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 3) + (((((int)threadIdx.x) * 11) + 10) / 24)) < 24) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 10) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 566) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 10))] = kernel[((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 10) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 10) % 24) * 3)))];
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.x))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 7))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 28))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 35))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 7))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 14))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 35))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 42))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 14))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 21))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 42))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 49))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 56))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 84))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 70))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 98))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 70))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 77))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 98))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 105))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 112))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 140))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 147))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 147))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 154))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 133))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 154))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 161))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 168))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 175))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 196))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 203))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 175))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 203))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 210))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 189))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 210))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 217))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    __syncthreads();
    pad_temp_shared[(((((int)threadIdx.z) * 28) + (((int)threadIdx.x) * 4)))] = (((1 <= ((((int)blockIdx.y) * 2) + ((((int)threadIdx.x) * 4) / 7))) && (((((int)blockIdx.y) * 2) + ((((int)threadIdx.x) * 4) / 7)) < 15)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (((((int)threadIdx.x) * 4) / 7) * 14)) + (((int)blockIdx.x) * 7)) + ((((int)threadIdx.x) * 4) % 7)) - 14))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 28) + (((int)threadIdx.x) * 4)) + 1))] = (((1 <= ((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 4) + 1) / 7))) && (((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 4) + 1) / 7)) < 15)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.x) * 4) + 1) / 7) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 1) % 7)) - 14))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 28) + (((int)threadIdx.x) * 4)) + 2))] = (((1 <= ((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 4) + 2) / 7))) && (((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 4) + 2) / 7)) < 15)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.x) * 4) + 2) / 7) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 2) % 7)) - 14))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 28) + (((int)threadIdx.x) * 4)) + 3))] = (((1 <= ((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 4) + 3) / 7))) && (((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 4) + 3) / 7)) < 15)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.x) * 4) + 3) / 7) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 3) % 7)) - 14))] : 0.000000e+00f);
    kernel_shared[(((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)))] = kernel[(((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + (((((int)threadIdx.x) * 11) / 24) * 1152)) + (rc_outer * 72)) + (((((int)threadIdx.x) * 11) % 24) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 1))] = kernel[(((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 1) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 1) % 24) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 2))] = kernel[(((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 2) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 2) % 24) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 3))] = kernel[(((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 3) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 3) % 24) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 4))] = kernel[(((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 4) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 4) % 24) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 5))] = kernel[(((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 5) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 5) % 24) * 3)) + 1))];
    if (((((int)threadIdx.z) * 3) + (((((int)threadIdx.x) * 11) + 6) / 24)) < 24) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 190) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 570) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 6))] = kernel[(((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 6) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 6) % 24) * 3)) + 1))];
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 3) + (((((int)threadIdx.x) * 11) + 7) / 24)) < 24) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 7) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 569) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 7))] = kernel[(((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 7) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 7) % 24) * 3)) + 1))];
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 3) + (((((int)threadIdx.x) * 11) + 8) / 24)) < 24) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 8) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 568) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 8))] = kernel[(((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 8) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 8) % 24) * 3)) + 1))];
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 3) + (((((int)threadIdx.x) * 11) + 9) / 24)) < 24) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 189) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 567) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 9))] = kernel[(((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 9) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 9) % 24) * 3)) + 1))];
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 3) + (((((int)threadIdx.x) * 11) + 10) / 24)) < 24) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 10) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 566) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 10))] = kernel[(((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 10) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 10) % 24) * 3)) + 1))];
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.x))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 7))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 28))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 35))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 7))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 14))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 35))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 42))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 14))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 21))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 42))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 49))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 56))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 84))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 70))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 98))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 70))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 77))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 98))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 105))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 112))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 140))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 147))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 147))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 154))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 133))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 154))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 161))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 168))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 175))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 196))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 203))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 175))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 203))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 210))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 189))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 210))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 217))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    __syncthreads();
    pad_temp_shared[(((((int)threadIdx.z) * 28) + (((int)threadIdx.x) * 4)))] = ((((1 <= ((((int)blockIdx.y) * 2) + ((((int)threadIdx.x) * 4) / 7))) && (((((int)blockIdx.y) * 2) + ((((int)threadIdx.x) * 4) / 7)) < 15)) && (((((int)blockIdx.x) * 7) + ((((int)threadIdx.x) * 4) % 7)) < 13)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + (((((int)threadIdx.x) * 4) / 7) * 14)) + (((int)blockIdx.x) * 7)) + ((((int)threadIdx.x) * 4) % 7)) - 13))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 28) + (((int)threadIdx.x) * 4)) + 1))] = ((((1 <= ((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 4) + 1) / 7))) && (((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 4) + 1) / 7)) < 15)) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 1) % 7)) < 13)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.x) * 4) + 1) / 7) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 1) % 7)) - 13))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 28) + (((int)threadIdx.x) * 4)) + 2))] = ((((1 <= ((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 4) + 2) / 7))) && (((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 4) + 2) / 7)) < 15)) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 2) % 7)) < 13)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.x) * 4) + 2) / 7) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 2) % 7)) - 13))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 28) + (((int)threadIdx.x) * 4)) + 3))] = ((((1 <= ((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 4) + 3) / 7))) && (((((int)blockIdx.y) * 2) + (((((int)threadIdx.x) * 4) + 3) / 7)) < 15)) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 3) % 7)) < 13)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 28)) + ((((((int)threadIdx.x) * 4) + 3) / 7) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 3) % 7)) - 13))] : 0.000000e+00f);
    kernel_shared[(((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)))] = kernel[(((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + (((((int)threadIdx.x) * 11) / 24) * 1152)) + (rc_outer * 72)) + (((((int)threadIdx.x) * 11) % 24) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 1))] = kernel[(((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 1) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 1) % 24) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 2))] = kernel[(((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 2) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 2) % 24) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 3))] = kernel[(((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 3) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 3) % 24) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 4))] = kernel[(((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 4) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 4) % 24) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 5))] = kernel[(((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 5) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 5) % 24) * 3)) + 2))];
    if (((((int)threadIdx.z) * 3) + (((((int)threadIdx.x) * 11) + 6) / 24)) < 24) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 190) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 570) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 6))] = kernel[(((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 6) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 6) % 24) * 3)) + 2))];
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 3) + (((((int)threadIdx.x) * 11) + 7) / 24)) < 24) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 7) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 569) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 7))] = kernel[(((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 7) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 7) % 24) * 3)) + 2))];
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 3) + (((((int)threadIdx.x) * 11) + 8) / 24)) < 24) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 8) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 568) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 8))] = kernel[(((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 8) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 8) % 24) * 3)) + 2))];
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 3) + (((((int)threadIdx.x) * 11) + 9) / 24)) < 24) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 189) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 567) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 9))] = kernel[(((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 9) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 9) % 24) * 3)) + 2))];
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 3) + (((((int)threadIdx.x) * 11) + 10) / 24)) < 24) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 10) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 566) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 10))] = kernel[(((((((((int)blockIdx.z) * 27648) + (((int)threadIdx.z) * 3456)) + ((((((int)threadIdx.x) * 11) + 10) / 24) * 1152)) + (rc_outer * 72)) + ((((((int)threadIdx.x) * 11) + 10) % 24) * 3)) + 2))];
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.x))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 7))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 28))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 35))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 7))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 14))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 35))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 42))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 14))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 21))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 42))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 49))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 56))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 84))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 70))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 98))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 70))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 77))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 98))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 105))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 112))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 140))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 147))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 147))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 154))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 133))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 154))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 161))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 168))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 175))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 196))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 203))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 175))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 203))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 210))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 189))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 210))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((int)threadIdx.x) + 217))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(5)]));
  }
  compute[((((((((int)blockIdx.z) * 4704) + (((int)threadIdx.z) * 588)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)))] = compute_local[(0)];
  compute[(((((((((int)blockIdx.z) * 4704) + (((int)threadIdx.z) * 588)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)) + 14))] = compute_local[(1)];
  compute[(((((((((int)blockIdx.z) * 4704) + (((int)threadIdx.z) * 588)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)) + 196))] = compute_local[(2)];
  compute[(((((((((int)blockIdx.z) * 4704) + (((int)threadIdx.z) * 588)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)) + 210))] = compute_local[(3)];
  compute[(((((((((int)blockIdx.z) * 4704) + (((int)threadIdx.z) * 588)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)) + 392))] = compute_local[(4)];
  compute[(((((((((int)blockIdx.z) * 4704) + (((int)threadIdx.z) * 588)) + (((int)blockIdx.y) * 28)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)) + 406))] = compute_local[(5)];
}





float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
    #pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
void pad_input(float * x, float *y){
    #pragma omp parallel for
    for(unsigned int i=0;i<(H + 2)*(W+2)*C;++i){
        y[i] = 0.0f;
    }
    #pragma omp parallel for
    for(unsigned int c=0;c<C;++c){
        for(unsigned int h=0;h<H;++h){
            for(unsigned int w=0;w<W;++w){
                unsigned int h_padded = h + 1;
                unsigned int w_padded = w + 1;
                y[c*(H+2)*(W+2) + h_padded*(W+2) + w_padded] = x[c*(H)*(W) + h*(W) + w];
            }
        }
    }
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float * padded_input = new float[C*(H+2)*(W+2)];
    pad_input(input, padded_input);
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);


    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);

    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);

    dim3 grid(2,7,4);
    dim3 block(7,1,8);

    float * paddedInputDevice;
    chkerr(hipMalloc(&paddedInputDevice, C * (H + 2) * (W + 2) * sizeof(float)));
    chkerr(hipMemcpy(paddedInputDevice, padded_input, C * (H + 2) * (W + 2) * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<endl;
    return 0;
}


