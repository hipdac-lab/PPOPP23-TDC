#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>

#define C 64
#define N 32
#define H 56
#define W 56

#define R 3
#define S 3
using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[7];
  __shared__ float pad_temp_shared[928];
  __shared__ float kernel_shared[288];
  float pad_temp_shared_local[54];
  float kernel_shared_local[18];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)))] = (((((1 <= ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + ((((int)threadIdx.y) * 15) >> 2)) % 58)) && (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + ((((int)threadIdx.y) * 15) >> 2)) % 58) < 57)) && (1 <= ((((int)blockIdx.x) * 2) + ((((int)threadIdx.y) * 15) & 3)))) && (((((int)blockIdx.x) * 2) + ((((int)threadIdx.y) * 15) & 3)) < 57)) ? data[(((((((rc_outer * 12544) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + ((((int)threadIdx.y) * 15) >> 2)) / 58) * 3136)) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + ((((int)threadIdx.y) * 15) >> 2)) % 58) * 56)) + (((int)blockIdx.x) * 2)) + ((((int)threadIdx.y) * 15) & 3)) - 57))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) + 1))] = (((((1 <= ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 1) >> 2)) % 58)) && (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 1) >> 2)) % 58) < 57)) && (1 <= ((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 1) & 3)))) && (((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 1) & 3)) < 57)) ? data[(((((((rc_outer * 12544) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 1) >> 2)) / 58) * 3136)) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 1) >> 2)) % 58) * 56)) + (((int)blockIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 1) & 3)) - 57))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) + 2))] = (((((1 <= ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 2) >> 2)) % 58)) && (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 2) >> 2)) % 58) < 57)) && (1 <= ((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 2) & 3)))) && (((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 2) & 3)) < 57)) ? data[(((((((rc_outer * 12544) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 2) >> 2)) / 58) * 3136)) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 2) >> 2)) % 58) * 56)) + (((int)blockIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 2) & 3)) - 57))] : 0.000000e+00f);
    if ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 3) >> 2)) < 232) {
      if ((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) < 925) {
        if (((((int)threadIdx.y) * 15) + (((int)threadIdx.x) * 8)) < 113) {
          pad_temp_shared[(((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) + 3))] = (((((1 <= ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 3) >> 2)) % 58)) && (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 3) >> 2)) % 58) < 57)) && (1 <= ((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 3) & 3)))) && (((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 3) & 3)) < 57)) ? data[(((((((rc_outer * 12544) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 3) >> 2)) / 58) * 3136)) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 3) >> 2)) % 58) * 56)) + (((int)blockIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 3) & 3)) - 57))] : 0.000000e+00f);
        }
      }
    }
    if ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + ((((int)threadIdx.y) * 15) >> 2)) < 231) {
      if ((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) < 924) {
        if (((((int)threadIdx.y) * 15) + (((int)threadIdx.x) * 8)) < 112) {
          pad_temp_shared[(((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) + 4))] = (((((1 <= (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + ((((int)threadIdx.y) * 15) >> 2)) + 1) % 58)) && ((((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + ((((int)threadIdx.y) * 15) >> 2)) + 1) % 58) < 57)) && (1 <= ((((int)blockIdx.x) * 2) + ((((int)threadIdx.y) * 15) & 3)))) && (((((int)blockIdx.x) * 2) + ((((int)threadIdx.y) * 15) & 3)) < 57)) ? data[(((((((rc_outer * 12544) + ((((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + ((((int)threadIdx.y) * 15) >> 2)) + 1) / 58) * 3136)) + ((((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + ((((int)threadIdx.y) * 15) >> 2)) + 1) % 58) * 56)) + (((int)blockIdx.x) * 2)) + ((((int)threadIdx.y) * 15) & 3)) - 57))] : 0.000000e+00f);
        }
      }
    }
    if ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 5) >> 2)) < 232) {
      if ((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) < 923) {
        if (((((int)threadIdx.y) * 15) + (((int)threadIdx.x) * 8)) < 111) {
          pad_temp_shared[(((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) + 5))] = (((((1 <= ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 5) >> 2)) % 58)) && (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 5) >> 2)) % 58) < 57)) && (1 <= ((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 1) & 3)))) && (((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 1) & 3)) < 57)) ? data[(((((((rc_outer * 12544) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 5) >> 2)) / 58) * 3136)) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 5) >> 2)) % 58) * 56)) + (((int)blockIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 1) & 3)) - 57))] : 0.000000e+00f);
        }
      }
    }
    if ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 6) >> 2)) < 232) {
      if ((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) < 922) {
        if (((((int)threadIdx.y) * 15) + (((int)threadIdx.x) * 8)) < 110) {
          pad_temp_shared[(((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) + 6))] = (((((1 <= ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 6) >> 2)) % 58)) && (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 6) >> 2)) % 58) < 57)) && (1 <= ((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 2) & 3)))) && (((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 2) & 3)) < 57)) ? data[(((((((rc_outer * 12544) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 6) >> 2)) / 58) * 3136)) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 6) >> 2)) % 58) * 56)) + (((int)blockIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 2) & 3)) - 57))] : 0.000000e+00f);
        }
      }
    }
    if ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 7) >> 2)) < 232) {
      if ((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) < 921) {
        if (((((int)threadIdx.y) * 15) + (((int)threadIdx.x) * 8)) < 109) {
          if (((int)threadIdx.x) < 1) {
            pad_temp_shared[(((((((int)threadIdx.z) * 116) + (((int)threadIdx.y) * 15)) + (((int)threadIdx.x) * 8)) + 7))] = (((((1 <= ((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 7) >> 2)) % 58)) && (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 7) >> 2)) % 58) < 57)) && (1 <= ((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 3) & 3)))) && (((((int)blockIdx.x) * 2) + (((((int)threadIdx.y) * 15) + 3) & 3)) < 57)) ? data[(((((((rc_outer * 12544) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 7) >> 2)) / 58) * 3136)) + (((((((int)threadIdx.z) * 29) + (((int)threadIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 7) >> 2)) % 58) * 56)) + (((int)blockIdx.x) * 2)) + (((((int)threadIdx.y) * 15) + 3) & 3)) - 57))] : 0.000000e+00f);
          }
        }
      }
    }
    if ((((((((int)threadIdx.y) * 5) / 3) + ((int)threadIdx.x)) / 12) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 4) + ((((((int)threadIdx.y) * 5) / 3) + ((int)threadIdx.x)) / 3)) < 32) {
        if ((((((int)threadIdx.z) * 12) + ((((int)threadIdx.y) * 5) / 3)) + ((int)threadIdx.x)) < 96) {
          if ((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) < 288) {
            if (((((int)threadIdx.y) * 5) + (((int)threadIdx.x) * 3)) < 36) {
              if ((((((int)blockIdx.z) * 8) + ((((((int)threadIdx.y) * 5) / 3) + ((int)threadIdx.x)) / 12)) + ((int)threadIdx.z)) < 32) {
                kernel_shared[((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((((((int)threadIdx.y) * 5) / 3) + ((int)threadIdx.x)) / 12) * 576)) + (((int)threadIdx.z) * 576)) + (rc_outer * 36)) + (((((((int)threadIdx.y) * 5) / 3) + ((int)threadIdx.x)) % 12) * 3)) + ((((int)threadIdx.y) * 5) % 3)))];
              }
            }
          }
        }
      }
    }
    if (((((((((int)threadIdx.y) * 5) + 1) / 3) + ((int)threadIdx.x)) / 12) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 4) + (((((((int)threadIdx.y) * 5) + 1) / 3) + ((int)threadIdx.x)) / 3)) < 32) {
        if ((((((int)threadIdx.z) * 12) + (((((int)threadIdx.y) * 5) + 1) / 3)) + ((int)threadIdx.x)) < 96) {
          if ((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) < 287) {
            if (((((int)threadIdx.y) * 5) + (((int)threadIdx.x) * 3)) < 35) {
              if ((((((int)blockIdx.z) * 8) + (((((((int)threadIdx.y) * 5) + 1) / 3) + ((int)threadIdx.x)) / 12)) + ((int)threadIdx.z)) < 32) {
                kernel_shared[(((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) + 1))] = kernel[(((((((((int)blockIdx.z) * 4608) + ((((((((int)threadIdx.y) * 5) + 1) / 3) + ((int)threadIdx.x)) / 12) * 576)) + (((int)threadIdx.z) * 576)) + (rc_outer * 36)) + ((((((((int)threadIdx.y) * 5) + 1) / 3) + ((int)threadIdx.x)) % 12) * 3)) + (((((int)threadIdx.y) * 5) + 1) % 3)))];
              }
            }
          }
        }
      }
    }
    if (((((((((int)threadIdx.y) * 5) + 2) / 3) + ((int)threadIdx.x)) / 12) + ((int)threadIdx.z)) < 8) {
      if (((((int)threadIdx.z) * 4) + (((((((int)threadIdx.y) * 5) + 2) / 3) + ((int)threadIdx.x)) / 3)) < 32) {
        if ((((((int)threadIdx.z) * 12) + (((((int)threadIdx.y) * 5) + 2) / 3)) + ((int)threadIdx.x)) < 96) {
          if ((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) < 286) {
            if (((((int)threadIdx.y) * 5) + (((int)threadIdx.x) * 3)) < 34) {
              if (((int)threadIdx.x) < 1) {
                if ((((((int)blockIdx.z) * 8) + (((((((int)threadIdx.y) * 5) + 2) / 3) + ((int)threadIdx.x)) / 12)) + ((int)threadIdx.z)) < 32) {
                  kernel_shared[(((((((int)threadIdx.z) * 36) + (((int)threadIdx.y) * 5)) + (((int)threadIdx.x) * 3)) + 2))] = kernel[(((((((((int)blockIdx.z) * 4608) + ((((((((int)threadIdx.y) * 5) + 2) / 3) + ((int)threadIdx.x)) / 12) * 576)) + (((int)threadIdx.z) * 576)) + (rc_outer * 36)) + ((((((((int)threadIdx.y) * 5) + 2) / 3) + ((int)threadIdx.x)) % 12) * 3)) + (((((int)threadIdx.y) * 5) + 2) % 3)))];
                }
              }
            }
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 28) + ((int)threadIdx.x)))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 1))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 2))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 4))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 5))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 6))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 8))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 9))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 10))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 12))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 13))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 14))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 16))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 17))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 18))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 20))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 21))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 22))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 24))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 25))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 26))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 28))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 29))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 30))];
    pad_temp_shared_local[(24)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 32))];
    pad_temp_shared_local[(25)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 33))];
    pad_temp_shared_local[(26)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 34))];
    pad_temp_shared_local[(27)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 232))];
    pad_temp_shared_local[(28)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 233))];
    pad_temp_shared_local[(29)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 234))];
    pad_temp_shared_local[(30)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 236))];
    pad_temp_shared_local[(31)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 237))];
    pad_temp_shared_local[(32)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 238))];
    pad_temp_shared_local[(33)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 240))];
    pad_temp_shared_local[(34)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 241))];
    pad_temp_shared_local[(35)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 242))];
    pad_temp_shared_local[(36)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 244))];
    pad_temp_shared_local[(37)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 245))];
    pad_temp_shared_local[(38)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 246))];
    pad_temp_shared_local[(39)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 248))];
    pad_temp_shared_local[(40)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 249))];
    pad_temp_shared_local[(41)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 250))];
    pad_temp_shared_local[(42)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 252))];
    pad_temp_shared_local[(43)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 253))];
    pad_temp_shared_local[(44)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 254))];
    pad_temp_shared_local[(45)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 256))];
    pad_temp_shared_local[(46)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 257))];
    pad_temp_shared_local[(47)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 258))];
    pad_temp_shared_local[(48)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 260))];
    pad_temp_shared_local[(49)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 261))];
    pad_temp_shared_local[(50)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 262))];
    pad_temp_shared_local[(51)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 264))];
    pad_temp_shared_local[(52)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 265))];
    pad_temp_shared_local[(53)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 266))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 36))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 36) + 1))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 36) + 2))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 36) + 3))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 36) + 4))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 36) + 5))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 36) + 6))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 36) + 7))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 36) + 8))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 36) + 9))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 36) + 10))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 36) + 11))];
    kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 36) + 12))];
    kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 36) + 13))];
    kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 36) + 14))];
    kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 36) + 15))];
    kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 36) + 16))];
    kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 36) + 17))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(1)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(1)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(1)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(2)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(2)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(3)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(4)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(4)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(4)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(5)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(5)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(5)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(6)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(6)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(6)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(6)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(24)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(7)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(7)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(7)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(7)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(25)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(8)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(8)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(8)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(8)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(26)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(27)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(30)] * kernel_shared_local[(9)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(33)] * kernel_shared_local[(9)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(36)] * kernel_shared_local[(9)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(39)] * kernel_shared_local[(9)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(42)] * kernel_shared_local[(9)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(45)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(28)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(31)] * kernel_shared_local[(10)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(34)] * kernel_shared_local[(10)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(37)] * kernel_shared_local[(10)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(40)] * kernel_shared_local[(10)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(43)] * kernel_shared_local[(10)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(46)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(29)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(32)] * kernel_shared_local[(11)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(35)] * kernel_shared_local[(11)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(38)] * kernel_shared_local[(11)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(41)] * kernel_shared_local[(11)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(44)] * kernel_shared_local[(11)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(47)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(30)] * kernel_shared_local[(12)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(33)] * kernel_shared_local[(12)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(36)] * kernel_shared_local[(12)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(39)] * kernel_shared_local[(12)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(42)] * kernel_shared_local[(12)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(45)] * kernel_shared_local[(12)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(48)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(31)] * kernel_shared_local[(13)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(34)] * kernel_shared_local[(13)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(37)] * kernel_shared_local[(13)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(40)] * kernel_shared_local[(13)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(43)] * kernel_shared_local[(13)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(46)] * kernel_shared_local[(13)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(49)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(32)] * kernel_shared_local[(14)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(35)] * kernel_shared_local[(14)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(38)] * kernel_shared_local[(14)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(41)] * kernel_shared_local[(14)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(44)] * kernel_shared_local[(14)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(47)] * kernel_shared_local[(14)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(50)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(33)] * kernel_shared_local[(15)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(36)] * kernel_shared_local[(15)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(39)] * kernel_shared_local[(15)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(42)] * kernel_shared_local[(15)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(45)] * kernel_shared_local[(15)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(48)] * kernel_shared_local[(15)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(51)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(34)] * kernel_shared_local[(16)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(37)] * kernel_shared_local[(16)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(40)] * kernel_shared_local[(16)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(43)] * kernel_shared_local[(16)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(46)] * kernel_shared_local[(16)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(49)] * kernel_shared_local[(16)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(52)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(35)] * kernel_shared_local[(17)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(38)] * kernel_shared_local[(17)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(41)] * kernel_shared_local[(17)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(44)] * kernel_shared_local[(17)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(47)] * kernel_shared_local[(17)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(50)] * kernel_shared_local[(17)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(53)] * kernel_shared_local[(17)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 464))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 465))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 466))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 468))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 469))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 470))];
    pad_temp_shared_local[(6)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 472))];
    pad_temp_shared_local[(7)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 473))];
    pad_temp_shared_local[(8)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 474))];
    pad_temp_shared_local[(9)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 476))];
    pad_temp_shared_local[(10)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 477))];
    pad_temp_shared_local[(11)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 478))];
    pad_temp_shared_local[(12)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 480))];
    pad_temp_shared_local[(13)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 481))];
    pad_temp_shared_local[(14)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 482))];
    pad_temp_shared_local[(15)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 484))];
    pad_temp_shared_local[(16)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 485))];
    pad_temp_shared_local[(17)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 486))];
    pad_temp_shared_local[(18)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 488))];
    pad_temp_shared_local[(19)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 489))];
    pad_temp_shared_local[(20)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 490))];
    pad_temp_shared_local[(21)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 492))];
    pad_temp_shared_local[(22)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 493))];
    pad_temp_shared_local[(23)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 494))];
    pad_temp_shared_local[(24)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 496))];
    pad_temp_shared_local[(25)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 497))];
    pad_temp_shared_local[(26)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 498))];
    pad_temp_shared_local[(27)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 696))];
    pad_temp_shared_local[(28)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 697))];
    pad_temp_shared_local[(29)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 698))];
    pad_temp_shared_local[(30)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 700))];
    pad_temp_shared_local[(31)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 701))];
    pad_temp_shared_local[(32)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 702))];
    pad_temp_shared_local[(33)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 704))];
    pad_temp_shared_local[(34)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 705))];
    pad_temp_shared_local[(35)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 706))];
    pad_temp_shared_local[(36)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 708))];
    pad_temp_shared_local[(37)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 709))];
    pad_temp_shared_local[(38)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 710))];
    pad_temp_shared_local[(39)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 712))];
    pad_temp_shared_local[(40)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 713))];
    pad_temp_shared_local[(41)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 714))];
    pad_temp_shared_local[(42)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 716))];
    pad_temp_shared_local[(43)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 717))];
    pad_temp_shared_local[(44)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 718))];
    pad_temp_shared_local[(45)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 720))];
    pad_temp_shared_local[(46)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 721))];
    pad_temp_shared_local[(47)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 722))];
    pad_temp_shared_local[(48)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 724))];
    pad_temp_shared_local[(49)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 725))];
    pad_temp_shared_local[(50)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 726))];
    pad_temp_shared_local[(51)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 728))];
    pad_temp_shared_local[(52)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 729))];
    pad_temp_shared_local[(53)] = pad_temp_shared[((((((int)threadIdx.y) * 28) + ((int)threadIdx.x)) + 730))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 36) + 18))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 36) + 19))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 36) + 20))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 36) + 21))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 36) + 22))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 36) + 23))];
    kernel_shared_local[(6)] = kernel_shared[(((((int)threadIdx.z) * 36) + 24))];
    kernel_shared_local[(7)] = kernel_shared[(((((int)threadIdx.z) * 36) + 25))];
    kernel_shared_local[(8)] = kernel_shared[(((((int)threadIdx.z) * 36) + 26))];
    kernel_shared_local[(9)] = kernel_shared[(((((int)threadIdx.z) * 36) + 27))];
    kernel_shared_local[(10)] = kernel_shared[(((((int)threadIdx.z) * 36) + 28))];
    kernel_shared_local[(11)] = kernel_shared[(((((int)threadIdx.z) * 36) + 29))];
    kernel_shared_local[(12)] = kernel_shared[(((((int)threadIdx.z) * 36) + 30))];
    kernel_shared_local[(13)] = kernel_shared[(((((int)threadIdx.z) * 36) + 31))];
    kernel_shared_local[(14)] = kernel_shared[(((((int)threadIdx.z) * 36) + 32))];
    kernel_shared_local[(15)] = kernel_shared[(((((int)threadIdx.z) * 36) + 33))];
    kernel_shared_local[(16)] = kernel_shared[(((((int)threadIdx.z) * 36) + 34))];
    kernel_shared_local[(17)] = kernel_shared[(((((int)threadIdx.z) * 36) + 35))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(1)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(1)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(1)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(1)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(1)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(2)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(2)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(2)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(2)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(2)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(3)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(3)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(3)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(3)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(3)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(4)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(4)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(4)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(4)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(4)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(5)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(5)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(5)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(5)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(5)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(5)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(6)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(6)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(6)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(6)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(6)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(24)] * kernel_shared_local[(6)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(7)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(7)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(7)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(7)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(7)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(25)] * kernel_shared_local[(7)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(8)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(8)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(8)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(8)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(8)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(26)] * kernel_shared_local[(8)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(27)] * kernel_shared_local[(9)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(30)] * kernel_shared_local[(9)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(33)] * kernel_shared_local[(9)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(36)] * kernel_shared_local[(9)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(39)] * kernel_shared_local[(9)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(42)] * kernel_shared_local[(9)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(45)] * kernel_shared_local[(9)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(28)] * kernel_shared_local[(10)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(31)] * kernel_shared_local[(10)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(34)] * kernel_shared_local[(10)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(37)] * kernel_shared_local[(10)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(40)] * kernel_shared_local[(10)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(43)] * kernel_shared_local[(10)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(46)] * kernel_shared_local[(10)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(29)] * kernel_shared_local[(11)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(32)] * kernel_shared_local[(11)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(35)] * kernel_shared_local[(11)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(38)] * kernel_shared_local[(11)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(41)] * kernel_shared_local[(11)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(44)] * kernel_shared_local[(11)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(47)] * kernel_shared_local[(11)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(30)] * kernel_shared_local[(12)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(33)] * kernel_shared_local[(12)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(36)] * kernel_shared_local[(12)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(39)] * kernel_shared_local[(12)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(42)] * kernel_shared_local[(12)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(45)] * kernel_shared_local[(12)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(48)] * kernel_shared_local[(12)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(31)] * kernel_shared_local[(13)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(34)] * kernel_shared_local[(13)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(37)] * kernel_shared_local[(13)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(40)] * kernel_shared_local[(13)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(43)] * kernel_shared_local[(13)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(46)] * kernel_shared_local[(13)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(49)] * kernel_shared_local[(13)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(32)] * kernel_shared_local[(14)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(35)] * kernel_shared_local[(14)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(38)] * kernel_shared_local[(14)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(41)] * kernel_shared_local[(14)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(44)] * kernel_shared_local[(14)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(47)] * kernel_shared_local[(14)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(50)] * kernel_shared_local[(14)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(33)] * kernel_shared_local[(15)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(36)] * kernel_shared_local[(15)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(39)] * kernel_shared_local[(15)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(42)] * kernel_shared_local[(15)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(45)] * kernel_shared_local[(15)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(48)] * kernel_shared_local[(15)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(51)] * kernel_shared_local[(15)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(34)] * kernel_shared_local[(16)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(37)] * kernel_shared_local[(16)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(40)] * kernel_shared_local[(16)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(43)] * kernel_shared_local[(16)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(46)] * kernel_shared_local[(16)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(49)] * kernel_shared_local[(16)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(52)] * kernel_shared_local[(16)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(35)] * kernel_shared_local[(17)]));
    compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(38)] * kernel_shared_local[(17)]));
    compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(41)] * kernel_shared_local[(17)]));
    compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(44)] * kernel_shared_local[(17)]));
    compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(47)] * kernel_shared_local[(17)]));
    compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(50)] * kernel_shared_local[(17)]));
    compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(53)] * kernel_shared_local[(17)]));
  }
  compute[((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)))] = compute_local[(0)];
  compute[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 56))] = compute_local[(1)];
  compute[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 112))] = compute_local[(2)];
  compute[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 168))] = compute_local[(3)];
  compute[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 224))] = compute_local[(4)];
  compute[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 280))] = compute_local[(5)];
  compute[(((((((((int)blockIdx.z) * 25088) + (((int)threadIdx.z) * 3136)) + (((int)threadIdx.y) * 392)) + (((int)blockIdx.x) * 2)) + ((int)threadIdx.x)) + 336))] = compute_local[(6)];
}





float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
    #pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
void pad_input(float * x, float *y){
    #pragma omp parallel for
    for(unsigned int i=0;i<(H + 2)*(W+2)*C;++i){
        y[i] = 0.0f;
    }
    #pragma omp parallel for
    for(unsigned int c=0;c<C;++c){
        for(unsigned int h=0;h<H;++h){
            for(unsigned int w=0;w<W;++w){
                unsigned int h_padded = h + 1;
                unsigned int w_padded = w + 1;
                y[c*(H+2)*(W+2) + h_padded*(W+2) + w_padded] = x[c*(H)*(W) + h*(W) + w];
            }
        }
    }
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float * padded_input = new float[C*(H+2)*(W+2)];
    pad_input(input, padded_input);
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);


    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);

    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);

    dim3 grid(28,1,4);
    dim3 block(2,8,8);

    float * paddedInputDevice;
    chkerr(hipMalloc(&paddedInputDevice, C * (H + 2) * (W + 2) * sizeof(float)));
    chkerr(hipMemcpy(paddedInputDevice, padded_input, C * (H + 2) * (W + 2) * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<endl;
    return 0;
}


