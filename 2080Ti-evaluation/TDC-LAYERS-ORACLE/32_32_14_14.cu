#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 2
#define TW 3
#define TC 4
#define C 32
#define N 32
#define H 14
#define W 14

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define WPAD (TWS*TW + 2)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[1];
  __shared__ float pad_temp_shared[216];
  __shared__ float kernel_shared[576];
  float pad_temp_shared_local[3];
  float kernel_shared_local[3];
  compute_local[(0)] = 0.000000e+00f;
  pad_temp_shared[(((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)))] = (((((1 <= (((((int)threadIdx.x) * 4) / 9) + ((int)blockIdx.y))) && ((((((int)threadIdx.x) * 4) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + ((((int)threadIdx.x) * 4) % 9)))) && (((((int)blockIdx.x) * 7) + ((((int)threadIdx.x) * 4) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + (((((int)threadIdx.x) * 4) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((((int)threadIdx.x) * 4) % 9)) - 15))] : 0.000000e+00f);
  pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 1))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 1) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 1) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 1) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 1) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 1) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 1) % 9)) - 15))] : 0.000000e+00f);
  pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 2))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 2) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 2) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 2) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 2) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 2) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 2) % 9)) - 15))] : 0.000000e+00f);
  if (((((((int)threadIdx.x) * 4) + 3) / 27) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 3) + (((((int)threadIdx.x) * 4) + 3) / 9)) < 24) {
      if (((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) < 213) {
        if (((int)threadIdx.x) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 3))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 3) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 3) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 3) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 3) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 3) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 3) % 9)) - 15))] : 0.000000e+00f);
        }
      }
    }
  }
  kernel_shared[(((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)))] = kernel[((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 1))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 1))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 2))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 2))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 3))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 3))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 4))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 4))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 5))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 5))];
  if (((((((int)threadIdx.x) * 11) + 6) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 6) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 190) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 570) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 6))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 6))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 7) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 7) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 7) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 569) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 7))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 7))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 8) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 8) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 8) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 568) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 8))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 8))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 9) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + ((((int)threadIdx.x) * 11) / 9)) < 63) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 189) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 567) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 9))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 9))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 10) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 10) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 10) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 566) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 10))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 10))];
          }
        }
      }
    }
  }
  __syncthreads();
  pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.x))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 9))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 18))];
  kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 1))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 10))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 19))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 2))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 11))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 20))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 27))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 36))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 45))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 28))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 37))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 46))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 29))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 38))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 47))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 54))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 72))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 55))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 64))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 73))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 56))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 65))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 74))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 81))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 90))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 99))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 82))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 100))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 83))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 92))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 101))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 108))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 117))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 109))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 118))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 127))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 110))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 128))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 135))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 144))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 153))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 136))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 145))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 154))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 137))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 146))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 155))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 162))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 171))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 180))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 163))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 172))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 181))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 164))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 173))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 189))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 198))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 207))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 190))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 199))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 208))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 191))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 200))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 209))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  __syncthreads();
  pad_temp_shared[(((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)))] = (((((1 <= (((((int)threadIdx.x) * 4) / 9) + ((int)blockIdx.y))) && ((((((int)threadIdx.x) * 4) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + ((((int)threadIdx.x) * 4) % 9)))) && (((((int)blockIdx.x) * 7) + ((((int)threadIdx.x) * 4) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + (((((int)threadIdx.x) * 4) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((((int)threadIdx.x) * 4) % 9)) + 1553))] : 0.000000e+00f);
  pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 1))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 1) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 1) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 1) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 1) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 1) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 1) % 9)) + 1553))] : 0.000000e+00f);
  pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 2))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 2) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 2) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 2) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 2) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 2) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 2) % 9)) + 1553))] : 0.000000e+00f);
  if (((((((int)threadIdx.x) * 4) + 3) / 27) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 3) + (((((int)threadIdx.x) * 4) + 3) / 9)) < 24) {
      if (((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) < 213) {
        if (((int)threadIdx.x) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 3))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 3) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 3) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 3) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 3) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 3) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 3) % 9)) + 1553))] : 0.000000e+00f);
        }
      }
    }
  }
  kernel_shared[(((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 72))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 1))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 73))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 2))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 74))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 3))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 75))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 4))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 76))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 5))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 77))];
  if (((((((int)threadIdx.x) * 11) + 6) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 6) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 190) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 570) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 6))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 78))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 7) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 7) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 7) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 569) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 7))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 79))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 8) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 8) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 8) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 568) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 8))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 80))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 9) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + ((((int)threadIdx.x) * 11) / 9)) < 63) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 189) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 567) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 9))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 81))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 10) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 10) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 10) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 566) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 10))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 82))];
          }
        }
      }
    }
  }
  __syncthreads();
  pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.x))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 9))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 18))];
  kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 1))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 10))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 19))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 2))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 11))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 20))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 27))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 36))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 45))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 28))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 37))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 46))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 29))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 38))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 47))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 54))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 72))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 55))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 64))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 73))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 56))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 65))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 74))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 81))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 90))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 99))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 82))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 100))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 83))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 92))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 101))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 108))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 117))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 109))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 118))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 127))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 110))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 128))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 135))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 144))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 153))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 136))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 145))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 154))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 137))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 146))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 155))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 162))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 171))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 180))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 163))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 172))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 181))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 164))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 173))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 189))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 198))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 207))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 190))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 199))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 208))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 191))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 200))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 209))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  __syncthreads();
  pad_temp_shared[(((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)))] = (((((1 <= (((((int)threadIdx.x) * 4) / 9) + ((int)blockIdx.y))) && ((((((int)threadIdx.x) * 4) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + ((((int)threadIdx.x) * 4) % 9)))) && (((((int)blockIdx.x) * 7) + ((((int)threadIdx.x) * 4) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + (((((int)threadIdx.x) * 4) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((((int)threadIdx.x) * 4) % 9)) + 3121))] : 0.000000e+00f);
  pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 1))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 1) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 1) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 1) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 1) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 1) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 1) % 9)) + 3121))] : 0.000000e+00f);
  pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 2))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 2) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 2) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 2) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 2) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 2) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 2) % 9)) + 3121))] : 0.000000e+00f);
  if (((((((int)threadIdx.x) * 4) + 3) / 27) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 3) + (((((int)threadIdx.x) * 4) + 3) / 9)) < 24) {
      if (((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) < 213) {
        if (((int)threadIdx.x) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 3))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 3) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 3) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 3) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 3) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 3) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 3) % 9)) + 3121))] : 0.000000e+00f);
        }
      }
    }
  }
  kernel_shared[(((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 144))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 1))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 145))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 2))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 146))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 3))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 147))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 4))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 148))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 5))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 149))];
  if (((((((int)threadIdx.x) * 11) + 6) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 6) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 190) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 570) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 6))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 150))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 7) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 7) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 7) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 569) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 7))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 151))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 8) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 8) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 8) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 568) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 8))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 152))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 9) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + ((((int)threadIdx.x) * 11) / 9)) < 63) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 189) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 567) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 9))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 153))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 10) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 10) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 10) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 566) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 10))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 154))];
          }
        }
      }
    }
  }
  __syncthreads();
  pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.x))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 9))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 18))];
  kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 1))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 10))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 19))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 2))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 11))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 20))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 27))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 36))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 45))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 28))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 37))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 46))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 29))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 38))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 47))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 54))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 72))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 55))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 64))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 73))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 56))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 65))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 74))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 81))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 90))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 99))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 82))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 100))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 83))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 92))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 101))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 108))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 117))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 109))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 118))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 127))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 110))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 128))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 135))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 144))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 153))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 136))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 145))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 154))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 137))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 146))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 155))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 162))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 171))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 180))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 163))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 172))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 181))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 164))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 173))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 189))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 198))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 207))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 190))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 199))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 208))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 191))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 200))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 209))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  __syncthreads();
  pad_temp_shared[(((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)))] = (((((1 <= (((((int)threadIdx.x) * 4) / 9) + ((int)blockIdx.y))) && ((((((int)threadIdx.x) * 4) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + ((((int)threadIdx.x) * 4) % 9)))) && (((((int)blockIdx.x) * 7) + ((((int)threadIdx.x) * 4) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + (((((int)threadIdx.x) * 4) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((((int)threadIdx.x) * 4) % 9)) + 4689))] : 0.000000e+00f);
  pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 1))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 1) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 1) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 1) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 1) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 1) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 1) % 9)) + 4689))] : 0.000000e+00f);
  pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 2))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 2) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 2) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 2) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 2) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 2) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 2) % 9)) + 4689))] : 0.000000e+00f);
  if (((((((int)threadIdx.x) * 4) + 3) / 27) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 3) + (((((int)threadIdx.x) * 4) + 3) / 9)) < 24) {
      if (((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) < 213) {
        if (((int)threadIdx.x) < 6) {
          pad_temp_shared[((((((int)threadIdx.z) * 27) + (((int)threadIdx.x) * 4)) + 3))] = (((((1 <= ((((((int)threadIdx.x) * 4) + 3) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 4) + 3) / 9) + ((int)blockIdx.y)) < 15)) && (1 <= ((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 3) % 9)))) && (((((int)blockIdx.x) * 7) + (((((int)threadIdx.x) * 4) + 3) % 9)) < 15)) ? data[(((((((((int)threadIdx.z) * 196) + ((((((int)threadIdx.x) * 4) + 3) / 9) * 14)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + (((((int)threadIdx.x) * 4) + 3) % 9)) + 4689))] : 0.000000e+00f);
        }
      }
    }
  }
  kernel_shared[(((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 216))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 1))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 217))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 2))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 218))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 3))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 219))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 4))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 220))];
  kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 5))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 221))];
  if (((((((int)threadIdx.x) * 11) + 6) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 6) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 190) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 570) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 6))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 222))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 7) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 7) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 7) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 569) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 7))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 223))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 8) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 8) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 8) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 568) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 8))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 224))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 9) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + ((((int)threadIdx.x) * 11) / 9)) < 63) {
      if (((((int)threadIdx.z) * 24) + ((((int)threadIdx.x) * 11) / 3)) < 189) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 567) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 9))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 225))];
          }
        }
      }
    }
  }
  if (((((((int)threadIdx.x) * 11) + 10) / 72) + ((int)threadIdx.z)) < 8) {
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 11) + 10) / 9)) < 64) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 11) + 10) / 3)) < 192) {
        if (((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) < 566) {
          if (((int)threadIdx.x) < 6) {
            kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.x) * 11)) + 10))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 288)) + (((int)threadIdx.x) * 11)) + 226))];
          }
        }
      }
    }
  }
  __syncthreads();
  pad_temp_shared_local[(0)] = pad_temp_shared[(((int)threadIdx.x))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 9))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 18))];
  kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 1))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 10))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 19))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 2))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 11))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 20))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 27))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 36))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 45))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 28))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 37))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 46))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 29))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 38))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 47))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 54))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 63))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 72))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 55))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 64))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 73))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 56))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 65))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 74))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 81))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 90))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 99))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 82))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 91))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 100))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 83))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 92))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 101))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 108))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 117))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 126))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 109))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 118))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 127))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 110))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 119))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 128))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 135))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 144))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 153))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 136))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 145))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 154))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 137))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 146))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 155))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 162))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 171))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 180))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 163))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 172))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 181))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 164))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 173))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 182))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 189))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 198))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 207))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 190))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 199))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 208))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  pad_temp_shared_local[(0)] = pad_temp_shared[((((int)threadIdx.x) + 191))];
  pad_temp_shared_local[(1)] = pad_temp_shared[((((int)threadIdx.x) + 200))];
  pad_temp_shared_local[(2)] = pad_temp_shared[((((int)threadIdx.x) + 209))];
  kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
  kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
  kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
  compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
  compute[((((((((int)blockIdx.z) * 1568) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 14)) + (((int)blockIdx.x) * 7)) + ((int)threadIdx.x)))] = compute_local[(0)];
}






class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_input_2_shared_memory(float *input, float *shared_input, unsigned int h_start,
                                           unsigned int h_end, unsigned int h_offset, unsigned int c_start,
                                           unsigned int warp_id, unsigned int lane_id, unsigned int warp_size){
    switch(h_offset){
        case 0:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + r * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
        case 1:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
                for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
                    unsigned int r = i/W;
                    unsigned int s = i%W;
                    shared_input[c*(TH + 2)*(WPAD) + (1 + r) * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
                }
            }
            break;
    }
}
__device__ __forceinline__ void switch_write_back(unsigned int write_h, unsigned int write_w, unsigned int h_out_start, unsigned int w_out_start, unsigned int n, float * outputs, float * temp_result){
	switch(write_h){
		case 1: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 3:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 3; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
		case 2: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 2:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 2; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
			case 3:
 			#pragma unroll
			for (unsigned int th = 0; th < 2; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 3; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
	}
}
__global__ void conv2d(float * __restrict__ input,const float * __restrict__ kernel, float * __restrict__ outputs){
    extern __shared__ float shared_input[];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / THS;
    const unsigned int th_id = tile_id % THS;
    const unsigned int tw_id = threadIdx.x / N;
    const int h_out_start = th_id * TH;
    const int w_out_start = tw_id * TW;
    const unsigned int warp_id = tw_id;
    const unsigned int lane_id = threadIdx.x % N;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    for(unsigned int i=threadIdx.x;i<TC*(TH+2)*WPAD;i+=blockDim.x){
        shared_input[i] = 0.0f;
    }
    unsigned int n = lane_id;
    unsigned int c_offset = tc_id * TC;
    int h_offset = (h_out_start == 0)?1:0;
    int h_padded_start = h_out_start;
    int h_padded_end = min(h_padded_start + TH + 2, H + 2);
    int h_non_padded_start = max(h_out_start - 1, 0);
    int h_non_padded_end = min(H, h_padded_end - 1);
    __syncthreads();
    load_input_2_shared_memory(input, shared_input, h_non_padded_start, h_non_padded_end, h_offset, c_offset, warp_id, lane_id, N);
    __syncthreads();
    #pragma unroll
    for(unsigned int c=0;c<TC;c++){
        #pragma unroll
        for(unsigned int r=0;r<R;++r){
            #pragma unroll
            for(unsigned int s=0;s<S;++s){
                data_array[r*S+s] = kernel[(c + c_offset)*N*9+r*3*N+s*N+n];
            }
        }
        		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[0];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 4]*data_array[2];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[0];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[0];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[1];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[3];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[1];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[2];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[2];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 4]*data_array[5];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[3];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[3];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[4];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[6];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[4];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[5];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[7];
		temp_result[1] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 3]*data_array[8];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[5];
		temp_result[2] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 4]*data_array[8];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 1]*data_array[6];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[6];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[7];
		temp_result[3] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 2]*data_array[8];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 3]*data_array[7];
		temp_result[4] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 3]*data_array[8];
		temp_result[5] += shared_input[c*(TH+2)*(WPAD) + 3 * WPAD + tw_id * TW + 4]*data_array[8];

    }
    switch_write_back(min(TH, H - h_out_start), min(TW, W - w_out_start), h_out_start, w_out_start, n, outputs, temp_result);
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(2,14,4);

        dim3 block(7,1,8);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    chkerr(hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d),hipFuncAttributeMaxDynamicSharedMemorySize, TC*(TH+2)*(WPAD)*4));
    hipEventRecord(event_start);
    conv2d<<<TCS*THS, N * TWS, TC*(TH+2)*(WPAD)*4>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/2080Ti-layers-eval-oracle.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<
                                   cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<","<<difference<<endl;
    return 0;
}


