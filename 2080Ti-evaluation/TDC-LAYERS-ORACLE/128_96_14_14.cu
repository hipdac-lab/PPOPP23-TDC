#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 2
#define TW 4
#define TC 16
#define C 128
#define N 96
#define H 14
#define W 14

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[2];
  __shared__ float pad_temp_shared[768];
  __shared__ float kernel_shared[864];
  float pad_temp_shared_local[6];
  float kernel_shared_local[12];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[(((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) % 3)) < 15)) && (1 <= ((((int)threadIdx.x) * 19) & 15))) && (((((int)threadIdx.x) * 19) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) % 3) * 14)) + ((((int)threadIdx.x) * 19) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 1))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 1) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 1) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 1) & 15))) && ((((((int)threadIdx.x) * 19) + 1) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 1) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 1) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 1) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 2))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 2) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 2) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 2) & 15))) && ((((((int)threadIdx.x) * 19) + 2) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 2) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 2) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 2) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 3))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 3) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 3) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 3) & 15))) && ((((((int)threadIdx.x) * 19) + 3) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 3) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 3) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 3) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 4))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 4) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 4) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 4) & 15))) && ((((((int)threadIdx.x) * 19) + 4) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 4) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 4) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 4) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 5))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 5) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 5) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 5) & 15))) && ((((((int)threadIdx.x) * 19) + 5) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 5) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 5) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 5) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 6))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 6) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 6) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 6) & 15))) && ((((((int)threadIdx.x) * 19) + 6) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 6) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 6) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 6) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 7))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 7) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 7) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 7) & 15))) && ((((((int)threadIdx.x) * 19) + 7) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 7) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 7) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 7) & 15)) - 15))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 8))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 8) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 8) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 8) & 15))) && ((((((int)threadIdx.x) * 19) + 8) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 8) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 8) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 8) & 15)) - 15))] : 0.000000e+00f);
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 9) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 759) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 9))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 9) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 9) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 9) & 15))) && ((((((int)threadIdx.x) * 19) + 9) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 9) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 9) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 9) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 10) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 758) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 10))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 10) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 10) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 10) & 15))) && ((((((int)threadIdx.x) * 19) + 10) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 10) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 10) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 10) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 11) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 757) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 11))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 11) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 11) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 11) & 15))) && ((((((int)threadIdx.x) * 19) + 11) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 11) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 11) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 11) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 12) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 756) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 12))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 12) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 12) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 12) & 15))) && ((((((int)threadIdx.x) * 19) + 12) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 12) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 12) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 12) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 13) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 755) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 13))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 13) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 13) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 13) & 15))) && ((((((int)threadIdx.x) * 19) + 13) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 13) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 13) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 13) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 14) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 754) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 14))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 14) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 14) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 14) & 15))) && ((((((int)threadIdx.x) * 19) + 14) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 14) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 14) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 14) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 15) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 753) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 15))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 15) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 15) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 15) & 15))) && ((((((int)threadIdx.x) * 19) + 15) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 15) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 15) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 15) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) < 47) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 752) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 16))] = (((((1 <= (((int)blockIdx.y) + ((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) + 1) % 3))) && ((((int)blockIdx.y) + ((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) + 1) % 3)) < 15)) && (1 <= ((((int)threadIdx.x) * 19) & 15))) && (((((int)threadIdx.x) * 19) & 15) < 15)) ? data[(((((((rc_outer * 3136) + (((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) + 1) / 3) * 196)) + (((int)blockIdx.y) * 14)) + (((((((int)threadIdx.z) * 16) + ((((int)threadIdx.x) * 19) >> 4)) + 1) % 3) * 14)) + ((((int)threadIdx.x) * 19) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 17) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 751) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 17))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 17) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 17) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 1) & 15))) && ((((((int)threadIdx.x) * 19) + 1) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 17) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 17) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 1) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    if (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 18) >> 4)) < 48) {
      if (((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) < 750) {
        if (((int)threadIdx.x) < 13) {
          pad_temp_shared[((((((int)threadIdx.z) * 256) + (((int)threadIdx.x) * 19)) + 18))] = (((((1 <= (((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 18) >> 4)) % 3))) && ((((int)blockIdx.y) + (((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 18) >> 4)) % 3)) < 15)) && (1 <= (((((int)threadIdx.x) * 19) + 2) & 15))) && ((((((int)threadIdx.x) * 19) + 2) & 15) < 15)) ? data[(((((((rc_outer * 3136) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 18) >> 4)) / 3) * 196)) + (((int)blockIdx.y) * 14)) + ((((((int)threadIdx.z) * 16) + (((((int)threadIdx.x) * 19) + 18) >> 4)) % 3) * 14)) + (((((int)threadIdx.x) * 19) + 2) & 15)) - 15))] : 0.000000e+00f);
        }
      }
    }
    kernel_shared[(((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + (((((int)threadIdx.x) * 7) / 48) * 1152)) + (rc_outer * 144)) + (((((int)threadIdx.x) * 7) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 1))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + (((((int)threadIdx.x) * 7) / 48) * 1152)) + (rc_outer * 144)) + (((((int)threadIdx.x) * 7) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 2))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + (((((int)threadIdx.x) * 7) / 48) * 1152)) + (rc_outer * 144)) + (((((int)threadIdx.x) * 7) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 3))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 1) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 1) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 4))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 1) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 1) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 5))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 1) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 1) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 6))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 2) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 2) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 7))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 2) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 2) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 8))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 2) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 2) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 9))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 3) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 3) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 10))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 3) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 3) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 11))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 3) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 3) % 48) * 3)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 12))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 4) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 4) % 48) * 3)))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 13))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 4) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 4) % 48) * 3)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 14))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 4) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 4) % 48) * 3)) + 2))];
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 5) / 48)) < 6) {
      if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.x) * 7) + 5) / 3)) < 96) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 283) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 849) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 15))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 5) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 5) % 48) * 3)))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 5) / 48)) < 6) {
      if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.x) * 7) + 5) / 3)) < 96) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 283) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 848) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 16))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 5) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 5) % 48) * 3)) + 1))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 5) / 48)) < 6) {
      if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.x) * 7) + 5) / 3)) < 96) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 283) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 847) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 17))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 5) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 5) % 48) * 3)) + 2))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 6) / 48)) < 6) {
      if (((((int)threadIdx.z) * 32) + ((((int)threadIdx.x) * 7) / 3)) < 94) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 282) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 846) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 18))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 6) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 6) % 48) * 3)))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 6) / 48)) < 6) {
      if (((((int)threadIdx.z) * 32) + ((((int)threadIdx.x) * 7) / 3)) < 94) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 282) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 845) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 19))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 6) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 6) % 48) * 3)) + 1))];
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 2) + (((((int)threadIdx.x) * 7) + 6) / 48)) < 6) {
      if (((((int)threadIdx.z) * 32) + ((((int)threadIdx.x) * 7) / 3)) < 94) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 7)) < 282) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) < 844) {
            if (((int)threadIdx.x) < 13) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 21)) + 20))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 2304)) + ((((((int)threadIdx.x) * 7) + 6) / 48) * 1152)) + (rc_outer * 144)) + ((((((int)threadIdx.x) * 7) + 6) % 48) * 3)) + 2))];
            }
          }
        }
      }
    }
    __syncthreads();
    for (int rc_inner_outer = 0; rc_inner_outer < 8; ++rc_inner_outer) {
      pad_temp_shared_local[(0)] = pad_temp_shared[(((rc_inner_outer * 96) + ((int)threadIdx.x)))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 1))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 2))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 48))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 49))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 50))];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)))];
      kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 432))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 1))];
      kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 433))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 2))];
      kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 434))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 9))];
      kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 441))];
      kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 10))];
      kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 442))];
      kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 11))];
      kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 443))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 16))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 17))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 18))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 64))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 65))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 66))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 3))];
      kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 435))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 4))];
      kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 436))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 5))];
      kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 437))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 12))];
      kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 444))];
      kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 13))];
      kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 445))];
      kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 14))];
      kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 446))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 32))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 33))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 34))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 80))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 81))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 96) + ((int)threadIdx.x)) + 82))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 6))];
      kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 438))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 7))];
      kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 439))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 8))];
      kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 440))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 15))];
      kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 447))];
      kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 16))];
      kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 448))];
      kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 17))];
      kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 144) + (rc_inner_outer * 18)) + 449))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(6)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(7)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(8)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(9)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(10)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(11)]));
    }
  }
  compute[(((((((int)blockIdx.z) * 1176) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 14)) + ((int)threadIdx.x)))] = compute_local[(0)];
  compute[((((((((int)blockIdx.z) * 1176) + (((int)threadIdx.z) * 196)) + (((int)blockIdx.y) * 14)) + ((int)threadIdx.x)) + 588))] = compute_local[(1)];
}






class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_data_2_register(float *__restrict__ data_array, unsigned int c_index, const float * __restrict__ kernel, unsigned int n_id){
    for(unsigned int r=0;r<R;++r){
        for(unsigned int s=0;s<S;++s){
            data_array[r*S+s] = kernel[c_index*N*9+r*3*N+s*N+n_id];
        }
    }
}
__device__ void switch_function( unsigned int switch_condition,float *temp_kernel,float v,float *temp_result){
	switch (switch_condition) {
		case 0:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*4+(0-s)] += result;
				}
			}
		break;
		case 1:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*4+(1-s)] += result;
				}
			}
		break;
		case 2:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*4+(2-s)] += result;
				}
			}
		break;
		case 3:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*4+(3-s)] += result;
				}
			}
		break;
		case 4:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*4+(4-s)] += result;
				}
			}
		break;
		case 5:
			#pragma unroll
			for ( int r = 0; r < 1; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(0-r)*4+(5-s)] += result;
				}
			}
		break;
		case 6:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*4+(0-s)] += result;
				}
			}
		break;
		case 7:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*4+(1-s)] += result;
				}
			}
		break;
		case 8:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*4+(2-s)] += result;
				}
			}
		break;
		case 9:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*4+(3-s)] += result;
				}
			}
		break;
		case 10:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*4+(4-s)] += result;
				}
			}
		break;
		case 11:
			#pragma unroll
			for ( int r = 0; r < 2; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(1-r)*4+(5-s)] += result;
				}
			}
		break;
		case 12:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*4+(0-s)] += result;
				}
			}
		break;
		case 13:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*4+(1-s)] += result;
				}
			}
		break;
		case 14:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*4+(2-s)] += result;
				}
			}
		break;
		case 15:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*4+(3-s)] += result;
				}
			}
		break;
		case 16:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*4+(4-s)] += result;
				}
			}
		break;
		case 17:
			#pragma unroll
			for ( int r = 1; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(2-r)*4+(5-s)] += result;
				}
			}
		break;
		case 18:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 1; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*4+(0-s)] += result;
				}
			}
		break;
		case 19:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 2; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*4+(1-s)] += result;
				}
			}
		break;
		case 20:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*4+(2-s)] += result;
				}
			}
		break;
		case 21:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 0; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*4+(3-s)] += result;
				}
			}
		break;
		case 22:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 1; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*4+(4-s)] += result;
				}
			}
		break;
		case 23:
			#pragma unroll
			for ( int r = 2; r < 3; r++) {
				#pragma unroll
				for ( int s = 2; s < 3; s++) {
					float result = v * temp_kernel[r*S+s];
					temp_result[(3-r)*4+(5-s)] += result;
				}
			}
		break;

	}
}
__global__ void transform(float *matrix, float *matrix2){
    for(unsigned int global_id = blockIdx.x * blockDim.x + threadIdx.x;global_id<C*H*W;global_id+=gridDim.x * blockDim.x){
        const float v = matrix[global_id];
        unsigned int c = global_id / (H*W);
        unsigned int hw = global_id % (H*W);
        int h = (hw)/W+1;
        int w = (hw)%W+1;
        int th_start = min(h/TH,THS-1);
        int tw_start = min(w/TW,TWS-1);
        for(int tile_h_id = th_start;tile_h_id>=0;tile_h_id--){
            if((tile_h_id*TH+TH+2)<=h){
                break;
            }
            for(int tile_w_id = tw_start;tile_w_id>=0;tile_w_id--){
                if((tile_w_id*TW+TW+2)<=w){
                    break;
                }
                unsigned int tile_id = tile_h_id * TWS + tile_w_id;
                unsigned int abs_h = h - tile_h_id*TH;
                unsigned int abs_w = w - tile_w_id*TW;
                matrix2[c*THS*TWS*(TH+2)*(TW+2)+tile_id*(TH+2)*(TW+2)+abs_h*(TW+2)+abs_w] = v;
            }
        }
    }
}
__device__ void load_input_2_shared_memory(float *values,float *shared_input,unsigned int warp_id,unsigned int lane_id,
                                           unsigned int tile_id,unsigned int tile_c_id){
    for(unsigned int c_id=warp_id;c_id<TC&&tile_c_id+c_id<C;c_id+=blockDim.x/32){
        for(unsigned int id = lane_id;id<(TH+2)*(TW+2);id+=32){
            shared_input[c_id*(TH+2)*(TW+2)+id] = values[(tile_c_id+c_id)*(THS*TWS)*(TH+2)*(TW+2)+tile_id*(TH+2)*(TW+2)+id];
        }
    }
}
__global__ void conv2d(float * __restrict__ values,const float * __restrict__ kernel, float * __restrict__ outputs){
    __shared__ float input[TC*(TH+2)*(TW+2)];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / (THS * TWS);
    const unsigned int th_id = (tile_id - tc_id * (THS*TWS))/TWS;
    const unsigned int tw_id = (tile_id - tc_id * (THS*TWS))%TWS;
    const unsigned int h_start = th_id * TH;
    const unsigned int w_start = tw_id * TW;
    const unsigned int warp_id = threadIdx.x / 32;
    const unsigned int lane_id = threadIdx.x % 32;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    load_input_2_shared_memory(values,input,warp_id,lane_id,tile_id - tc_id * (THS*TWS),tc_id*TC);
    __syncthreads();
    float v;
    unsigned int n = threadIdx.x;
    unsigned int c_offset = tc_id * TC;
#pragma unroll
    for(unsigned int c=0;c<TC;c++){
        load_data_2_register(data_array,c + c_offset,kernel,n);
#pragma unroll
        for(unsigned int i=0;i<(TH+2)*(TW+2);++i){
            v = input[i + c*(TH+2)*(TW+2)];
            switch_function(i,data_array,v,temp_result);
        }
    }
#pragma unroll
    for (unsigned int th = 0; th < TH; ++th) {
#pragma unroll
        for (unsigned int tw = 0; tw < TW; ++tw) {
            if (h_start + th >= H || w_start + tw >= W) {
                continue;
            }
            atomicAdd(&outputs[n*H*W+(h_start + th) * W+(w_start + tw)],temp_result[(th * TW + tw)]);
        }
    }
}
float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(1,14,16);

        dim3 block(14,1,3);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    unsigned int blkDim = ((N - 1)/32 + 1) * 32;
    hipEventRecord(event_start);
    transform<<<216,1024>>>(device_input,matrix);
    conv2d<<<TCS*THS*TWS,blkDim>>>(matrix,device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/2080Ti-layers-eval-oracle.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_tvm, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<endl;
    return 0;
}


