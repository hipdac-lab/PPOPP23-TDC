#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>
#define TH 1
#define TW 1
#define TC 16
#define C 64
#define N 32
#define H 7
#define W 7

#define TCS ((C-1)/TC + 1)
#define THS ((H-1)/TH + 1)
#define TWS ((W-1)/TW+1)
#define WPAD (TWS*TW + 2)
#define R 3
#define S 3


using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[1];
  __shared__ float pad_temp_shared[864];
  __shared__ float kernel_shared[1152];
  float pad_temp_shared_local[24];
  float kernel_shared_local[24];
  compute_local[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 2; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[(((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)))] = (((((1 <= ((((((int)threadIdx.x) * 31) % 27) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 31) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= ((((int)threadIdx.x) * 31) % 9))) && (((((int)threadIdx.x) * 31) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + (((((int)threadIdx.x) * 31) / 27) * 49)) + ((((((int)threadIdx.x) * 31) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + ((((int)threadIdx.x) * 31) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 1))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 1) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 1) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 1) % 9))) && ((((((int)threadIdx.x) * 31) + 1) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 1) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 1) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 1) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 2))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 2) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 2) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 2) % 9))) && ((((((int)threadIdx.x) * 31) + 2) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 2) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 2) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 2) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 3))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 3) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 3) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 3) % 9))) && ((((((int)threadIdx.x) * 31) + 3) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 3) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 3) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 3) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 4))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 4) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 4) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 4) % 9))) && ((((((int)threadIdx.x) * 31) + 4) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 4) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 4) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 4) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 5))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 5) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 5) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 5) % 9))) && ((((((int)threadIdx.x) * 31) + 5) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 5) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 5) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 5) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 6))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 6) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 6) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 6) % 9))) && ((((((int)threadIdx.x) * 31) + 6) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 6) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 6) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 6) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 7))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 7) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 7) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 7) % 9))) && ((((((int)threadIdx.x) * 31) + 7) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 7) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 7) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 7) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 8))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 8) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 8) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 8) % 9))) && ((((((int)threadIdx.x) * 31) + 8) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 8) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 8) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 8) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 9))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 9) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 9) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= ((((int)threadIdx.x) * 31) % 9))) && (((((int)threadIdx.x) * 31) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 9) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 9) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + ((((int)threadIdx.x) * 31) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 10))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 10) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 10) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 1) % 9))) && ((((((int)threadIdx.x) * 31) + 1) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 10) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 10) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 1) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 11))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 11) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 11) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 2) % 9))) && ((((((int)threadIdx.x) * 31) + 2) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 11) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 11) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 2) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 12))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 12) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 12) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 3) % 9))) && ((((((int)threadIdx.x) * 31) + 3) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 12) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 12) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 3) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 13))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 13) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 13) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 4) % 9))) && ((((((int)threadIdx.x) * 31) + 4) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 13) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 13) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 4) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 14))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 14) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 14) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 5) % 9))) && ((((((int)threadIdx.x) * 31) + 5) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 14) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 14) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 5) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 15))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 15) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 15) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 6) % 9))) && ((((((int)threadIdx.x) * 31) + 6) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 15) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 15) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 6) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 16))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 16) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 16) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 7) % 9))) && ((((((int)threadIdx.x) * 31) + 7) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 16) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 16) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 7) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 17))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 17) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 17) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 8) % 9))) && ((((((int)threadIdx.x) * 31) + 8) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 17) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 17) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 8) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 18))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 18) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 18) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= ((((int)threadIdx.x) * 31) % 9))) && (((((int)threadIdx.x) * 31) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 18) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 18) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + ((((int)threadIdx.x) * 31) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 19))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 19) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 19) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 1) % 9))) && ((((((int)threadIdx.x) * 31) + 1) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 19) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 19) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 1) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 20))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 20) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 20) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 2) % 9))) && ((((((int)threadIdx.x) * 31) + 2) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 20) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 20) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 2) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 21))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 21) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 21) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 3) % 9))) && ((((((int)threadIdx.x) * 31) + 3) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 21) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 21) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 3) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 22))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 22) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 22) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 4) % 9))) && ((((((int)threadIdx.x) * 31) + 4) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 22) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 22) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 4) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 23))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 23) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 23) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 5) % 9))) && ((((((int)threadIdx.x) * 31) + 5) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 23) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 23) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 5) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 24))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 24) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 24) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 6) % 9))) && ((((((int)threadIdx.x) * 31) + 6) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 24) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 24) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 6) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 25))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 25) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 25) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 7) % 9))) && ((((((int)threadIdx.x) * 31) + 7) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 25) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 25) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 7) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 26))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 26) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 26) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 8) % 9))) && ((((((int)threadIdx.x) * 31) + 8) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 26) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 26) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 8) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 27))] = (((((1 <= ((((((int)threadIdx.x) * 31) % 27) / 9) + ((int)blockIdx.y))) && (((((((int)threadIdx.x) * 31) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= ((((int)threadIdx.x) * 31) % 9))) && (((((int)threadIdx.x) * 31) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + (((((int)threadIdx.x) * 31) / 27) * 49)) + ((((((int)threadIdx.x) * 31) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + ((((int)threadIdx.x) * 31) % 9)) + 41))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 28))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 1) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 1) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 1) % 9))) && ((((((int)threadIdx.x) * 31) + 1) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 28) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 1) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 1) % 9)) - 8))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 29))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 2) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 2) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 2) % 9))) && ((((((int)threadIdx.x) * 31) + 2) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 29) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 2) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 2) % 9)) - 8))] : 0.000000e+00f);
    if (((((int)threadIdx.z) * 8) + (((((int)threadIdx.x) * 31) + 30) / 27)) < 32) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.x) * 31) + 30) / 9)) < 96) {
        if (((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) < 834) {
          if (((int)threadIdx.x) < 6) {
            pad_temp_shared[((((((int)threadIdx.z) * 216) + (((int)threadIdx.x) * 31)) + 30))] = (((((1 <= (((((((int)threadIdx.x) * 31) + 3) % 27) / 9) + ((int)blockIdx.y))) && ((((((((int)threadIdx.x) * 31) + 3) % 27) / 9) + ((int)blockIdx.y)) < 8)) && (1 <= (((((int)threadIdx.x) * 31) + 3) % 9))) && ((((((int)threadIdx.x) * 31) + 3) % 9) < 8)) ? data[((((((((rc_outer * 1568) + (((int)threadIdx.z) * 392)) + ((((((int)threadIdx.x) * 31) + 30) / 27) * 49)) + (((((((int)threadIdx.x) * 31) + 3) % 27) / 9) * 7)) + (((int)blockIdx.y) * 7)) + (((((int)threadIdx.x) * 31) + 3) % 9)) - 8))] : 0.000000e+00f);
          }
        }
      }
    }
    kernel_shared[(((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)))] = kernel[(((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 1))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 2))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 3))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 3))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 4))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 4))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 5))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 5))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 6))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 6))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 7))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 7))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 8))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 8))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 9))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 9))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 10))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 10))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 11))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 11))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 12))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 12))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 13))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 13))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 14))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 14))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 15))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 15))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 16))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 16))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 17))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 17))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 18))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 18))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 19))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 19))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 20))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 20))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 21))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 21))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 22))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 22))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 23))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 23))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 24))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 24))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 25))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 25))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 26))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 26))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 27))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 27))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 28))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 28))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 29))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 29))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 30))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 30))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 31))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 31))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 32))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 32))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 33))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 33))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 34))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 34))];
    kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 35))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 35))];
    if (((((((int)threadIdx.x) * 14) + 12) / 96) + ((int)threadIdx.z)) < 4) {
      if (((((int)threadIdx.z) * 32) + ((((int)threadIdx.x) * 14) / 3)) < 124) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 14)) < 372) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) < 1116) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 36))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 36))];
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 14) + 12) / 96) + ((int)threadIdx.z)) < 4) {
      if (((((int)threadIdx.z) * 32) + ((((int)threadIdx.x) * 14) / 3)) < 124) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 14)) < 372) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) < 1115) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 37))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 37))];
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 14) + 12) / 96) + ((int)threadIdx.z)) < 4) {
      if (((((int)threadIdx.z) * 32) + ((((int)threadIdx.x) * 14) / 3)) < 124) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 14)) < 372) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) < 1114) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 38))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 38))];
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 14) + 13) / 96) + ((int)threadIdx.z)) < 4) {
      if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.x) * 14) + 13) / 3)) < 128) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 14)) < 371) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) < 1113) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 39))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 39))];
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 14) + 13) / 96) + ((int)threadIdx.z)) < 4) {
      if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.x) * 14) + 13) / 3)) < 128) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 14)) < 371) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) < 1112) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 40))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 40))];
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.x) * 14) + 13) / 96) + ((int)threadIdx.z)) < 4) {
      if (((((int)threadIdx.z) * 32) + (((((int)threadIdx.x) * 14) + 13) / 3)) < 128) {
        if (((((int)threadIdx.z) * 96) + (((int)threadIdx.x) * 14)) < 371) {
          if (((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) < 1111) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 288) + (((int)threadIdx.x) * 42)) + 41))] = kernel[((((((((int)blockIdx.z) * 2304) + (((int)threadIdx.z) * 576)) + (rc_outer * 288)) + (((int)threadIdx.x) * 42)) + 41))];
            }
          }
        }
      }
    }
    __syncthreads();
    for (int rc_inner_outer = 0; rc_inner_outer < 4; ++rc_inner_outer) {
      pad_temp_shared_local[(0)] = pad_temp_shared[(((rc_inner_outer * 216) + ((int)threadIdx.x)))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 1))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 2))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 27))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 28))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 29))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 54))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 55))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 56))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 81))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 82))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 83))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 108))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 109))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 110))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 135))];
      pad_temp_shared_local[(16)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 136))];
      pad_temp_shared_local[(17)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 137))];
      pad_temp_shared_local[(18)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 162))];
      pad_temp_shared_local[(19)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 163))];
      pad_temp_shared_local[(20)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 164))];
      pad_temp_shared_local[(21)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 189))];
      pad_temp_shared_local[(22)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 190))];
      pad_temp_shared_local[(23)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 191))];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 1))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 2))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 9))];
      kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 10))];
      kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 11))];
      kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 18))];
      kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 19))];
      kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 20))];
      kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 27))];
      kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 28))];
      kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 29))];
      kernel_shared_local[(12)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 36))];
      kernel_shared_local[(13)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 37))];
      kernel_shared_local[(14)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 38))];
      kernel_shared_local[(15)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 45))];
      kernel_shared_local[(16)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 46))];
      kernel_shared_local[(17)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 47))];
      kernel_shared_local[(18)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 54))];
      kernel_shared_local[(19)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 55))];
      kernel_shared_local[(20)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 56))];
      kernel_shared_local[(21)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 63))];
      kernel_shared_local[(22)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 64))];
      kernel_shared_local[(23)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 65))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 9))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 10))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 11))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 36))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 37))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 38))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 63))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 64))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 65))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 90))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 91))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 92))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 117))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 118))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 119))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 144))];
      pad_temp_shared_local[(16)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 145))];
      pad_temp_shared_local[(17)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 146))];
      pad_temp_shared_local[(18)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 171))];
      pad_temp_shared_local[(19)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 172))];
      pad_temp_shared_local[(20)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 173))];
      pad_temp_shared_local[(21)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 198))];
      pad_temp_shared_local[(22)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 199))];
      pad_temp_shared_local[(23)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 200))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 3))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 4))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 5))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 12))];
      kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 13))];
      kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 14))];
      kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 21))];
      kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 22))];
      kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 23))];
      kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 30))];
      kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 31))];
      kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 32))];
      kernel_shared_local[(12)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 39))];
      kernel_shared_local[(13)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 40))];
      kernel_shared_local[(14)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 41))];
      kernel_shared_local[(15)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 48))];
      kernel_shared_local[(16)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 49))];
      kernel_shared_local[(17)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 50))];
      kernel_shared_local[(18)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 57))];
      kernel_shared_local[(19)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 58))];
      kernel_shared_local[(20)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 59))];
      kernel_shared_local[(21)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 66))];
      kernel_shared_local[(22)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 67))];
      kernel_shared_local[(23)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 68))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 18))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 19))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 20))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 45))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 46))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 47))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 72))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 73))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 74))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 99))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 100))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 101))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 126))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 127))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 128))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 153))];
      pad_temp_shared_local[(16)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 154))];
      pad_temp_shared_local[(17)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 155))];
      pad_temp_shared_local[(18)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 180))];
      pad_temp_shared_local[(19)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 181))];
      pad_temp_shared_local[(20)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 182))];
      pad_temp_shared_local[(21)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 207))];
      pad_temp_shared_local[(22)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 208))];
      pad_temp_shared_local[(23)] = pad_temp_shared[((((rc_inner_outer * 216) + ((int)threadIdx.x)) + 209))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 6))];
      kernel_shared_local[(1)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 7))];
      kernel_shared_local[(2)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 8))];
      kernel_shared_local[(3)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 15))];
      kernel_shared_local[(4)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 16))];
      kernel_shared_local[(5)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 17))];
      kernel_shared_local[(6)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 24))];
      kernel_shared_local[(7)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 25))];
      kernel_shared_local[(8)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 26))];
      kernel_shared_local[(9)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 33))];
      kernel_shared_local[(10)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 34))];
      kernel_shared_local[(11)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 35))];
      kernel_shared_local[(12)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 42))];
      kernel_shared_local[(13)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 43))];
      kernel_shared_local[(14)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 44))];
      kernel_shared_local[(15)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 51))];
      kernel_shared_local[(16)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 52))];
      kernel_shared_local[(17)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 53))];
      kernel_shared_local[(18)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 60))];
      kernel_shared_local[(19)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 61))];
      kernel_shared_local[(20)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 62))];
      kernel_shared_local[(21)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 69))];
      kernel_shared_local[(22)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 70))];
      kernel_shared_local[(23)] = kernel_shared[((((((int)threadIdx.z) * 288) + (rc_inner_outer * 72)) + 71))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(6)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(7)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(8)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(9)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(10)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(11)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(12)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(13)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(14)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(15)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(16)] * kernel_shared_local[(16)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(17)] * kernel_shared_local[(17)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(18)] * kernel_shared_local[(18)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(19)] * kernel_shared_local[(19)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(20)] * kernel_shared_local[(20)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(21)] * kernel_shared_local[(21)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(22)] * kernel_shared_local[(22)]));
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(23)] * kernel_shared_local[(23)]));
    }
  }
  compute[(((((((int)blockIdx.z) * 196) + (((int)threadIdx.z) * 49)) + (((int)blockIdx.y) * 7)) + ((int)threadIdx.x)))] = compute_local[(0)];
}






class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}

class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
__device__ void load_input_2_shared_memory(float *input, float *shared_input, unsigned int h_start,
                                           unsigned int h_end, unsigned int h_offset, unsigned int c_start,
                                           unsigned int warp_id, unsigned int lane_id, unsigned int warp_size){
    switch(h_offset){
        case 0:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
        for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
            unsigned int r = i/W;
            unsigned int s = i%W;
            shared_input[c*(TH + 2)*(WPAD) + r * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
        }
    }
            break;
        case 1:
            for(unsigned int c = warp_id; c<TC; c+=TWS){
        for(unsigned int i=lane_id; i<(h_end - h_start) * W; i+=warp_size){
            unsigned int r = i/W;
            unsigned int s = i%W;
            shared_input[c*(TH + 2)*(WPAD) + (1 + r) * WPAD + s + 1] = input[(c_start + c) * H * W + h_start * W + i];
        }
    }
            break;
    }
}
__device__ __forceinline__ void switch_write_back(unsigned int write_h, unsigned int write_w, unsigned int h_out_start, unsigned int w_out_start, unsigned int n, float * outputs, float * temp_result){
	switch(write_h){
		case 1: 
 		switch(write_w){
			case 1:
 			#pragma unroll
			for (unsigned int th = 0; th < 1; ++th) { 
				#pragma unroll
				for (unsigned int tw = 0; tw < 1; ++tw) { 
					atomicAdd(&outputs[n*H*W+(h_out_start + th) * W+(w_out_start + tw)],temp_result[(th * TW + tw)]);
				}
			}
			break;
		} 
		break;
	}
}
        __global__ void conv2d(float * __restrict__ input,const float * __restrict__ kernel, float * __restrict__ outputs){
    extern __shared__ float shared_input[];
    const unsigned int tile_id = blockIdx.x;
    const unsigned int tc_id = tile_id / THS;
    const unsigned int th_id = tile_id % THS;
    const unsigned int tw_id = threadIdx.x / N;
    const int h_out_start = th_id * TH;
    const int w_out_start = tw_id * TW;
    const unsigned int warp_id = tw_id;
    const unsigned int lane_id = threadIdx.x % N;
    float data_array[9];
    float temp_result[TH*TW] = {0.0f};
    for(unsigned int i=threadIdx.x;i<TC*(TH+2)*WPAD;i+=blockDim.x){
        shared_input[i] = 0.0f;
    }
    unsigned int n = lane_id;
    unsigned int c_offset = tc_id * TC;
    int h_offset = (h_out_start == 0)?1:0;
    int h_padded_start = h_out_start;
    int h_padded_end = min(h_padded_start + TH + 2, H + 2);
    int h_non_padded_start = max(h_out_start - 1, 0);
    int h_non_padded_end = min(H, h_padded_end - 1);
    __syncthreads();
    load_input_2_shared_memory(input, shared_input, h_non_padded_start, h_non_padded_end, h_offset, c_offset, warp_id, lane_id, N);
    __syncthreads();
#pragma unroll
    for(unsigned int c=0;c<TC;c++){
#pragma unroll
        for(unsigned int r=0;r<R;++r){
#pragma unroll
            for(unsigned int s=0;s<S;++s){
                data_array[r*S+s] = kernel[(c + c_offset)*N*9+r*3*N+s*N+n];
            }
        }
        		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 0]*data_array[0];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 1]*data_array[1];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 0 * WPAD + tw_id * TW + 2]*data_array[2];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 0]*data_array[3];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 1]*data_array[4];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 1 * WPAD + tw_id * TW + 2]*data_array[5];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 0]*data_array[6];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 1]*data_array[7];
		temp_result[0] += shared_input[c*(TH+2)*(WPAD) + 2 * WPAD + tw_id * TW + 2]*data_array[8];

    }
    switch_write_back(min(TH, H - h_out_start), min(TW, W - w_out_start), h_out_start, w_out_start, n, outputs, temp_result);
}

float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
#pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    float *matrix;
    hipMalloc(&matrix,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    hipMemset(matrix,0,C*(TH+2)*(TW+2)*THS*TWS*sizeof(float));
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);

    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);


    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);


        dim3 grid(1,7,8);

                dim3 block(7,1,4);

    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tvm;
    hipEventElapsedTime(&time_tvm, event_start, event_stop);
    float *out_tvm = new float[N*H*W];
    hipMemcpy(out_tvm,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    hipMemset(device_out, 0, sizeof(float)*N*H*W);

    chkerr(hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d),hipFuncAttributeMaxDynamicSharedMemorySize, TC*(TH+2)*(WPAD)*4));
    hipEventRecord(event_start);
    conv2d<<<TCS*THS, N * TWS, TC*(TH+2)*(WPAD)*4>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    ofstream outfile;
    char buffer[1000];
    int ret = sprintf(buffer,"%d,%d,%d,%d,%f,%f,%f,%f,%f,%f,%f,%f,%f\n",N,C,H,W,
            cudnnFFTTime,cudnnWinogradeTimeNon,cudnnGemmTime,time_tvm,time_tdc,
            cudnnFFTTime/time_tdc,cudnnWinogradeTimeNon/time_tdc,cudnnGemmTime/time_tdc,time_tvm/time_tdc);
    outfile.open("../../evaluation_outcome/A100-layers-eval-modeling.csv", std::ios_base::app);
    outfile << buffer;


    float difference = check_diff(out_tvm, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<
                                   time_tvm<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<
                                   cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<","<<time_tvm/time_tdc<<","<<difference<<endl;
    return 0;
}


