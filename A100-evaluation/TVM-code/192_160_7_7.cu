#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>

#define C 192
#define N 160
#define H 7
#define W 7

#define R 3
#define S 3
using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[1];
  __shared__ float pad_temp_shared[1512];
  __shared__ float kernel_shared[288];
  float pad_temp_shared_local[6];
  float kernel_shared_local[6];
  compute_local[(0)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)))] = ((((7 <= (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 63)) && ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 63) < 56)) && (1 <= (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 7))) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) / 63) * 49)) + (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 63)) - 8))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 1))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) % 63) < 56)) && (1 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) % 7))) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) % 63)) - 8))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 2))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) % 63) < 56)) && (1 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) % 7))) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) % 63)) - 8))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 3))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) % 63) < 56)) && (1 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) % 7))) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) % 63)) - 8))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 4))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) % 63) < 56)) && (1 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) % 7))) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) % 63)) - 8))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 5))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) % 63) < 56)) && (1 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) % 7))) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) % 63)) - 8))] : 0.000000e+00f);
    if (((((int)threadIdx.z) * 6) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) / 63)) < 24) {
      if (((((int)threadIdx.z) * 54) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) / 7)) < 216) {
        if ((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) < 1506) {
          if (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) < 372) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 6))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) % 63) < 56)) && (1 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) % 7))) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) % 63)) - 8))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 6) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) / 63)) < 24) {
      if (((((int)threadIdx.z) * 54) + (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) / 7)) < 215) {
        if ((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) < 1505) {
          if (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) < 371) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 7))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) % 63) < 56)) && (1 <= (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 7))) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) % 63)) - 8))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) / 72) + ((int)threadIdx.z)) < 4) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) / 3)) < 96) {
        if ((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)) < 288) {
          if (((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) < 72) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)))] = kernel[((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + (((int)threadIdx.x) * 6)))];
            }
          }
        }
      }
    }
    if ((((((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) + 1) / 72) + ((int)threadIdx.z)) < 4) {
      if (((((int)threadIdx.z) * 24) + ((((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) + 1) / 3)) < 96) {
        if ((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)) < 287) {
          if (((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) < 71) {
            if (((int)threadIdx.x) < 5) {
              kernel_shared[(((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)) + 1))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + (((int)threadIdx.x) * 6)) + 3))];
            }
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 7))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 14))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 63))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 70))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 77))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 126))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 133))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 140))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 189))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 196))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 203))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 252))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 259))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 266))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 315))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 322))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 329))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 378))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 385))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 392))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 441))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 448))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 455))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 504))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 511))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 518))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 567))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 574))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 581))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 630))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 637))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 644))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 693))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 700))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 707))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 756))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 763))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 770))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 819))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 826))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 833))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 882))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 889))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 896))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 945))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 952))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 959))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1008))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1015))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1022))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1071))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1078))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1085))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1134))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1141))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1148))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1197))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1204))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1211))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1260))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1267))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1274))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1323))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1330))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1337))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1386))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1393))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1400))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1449))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1456))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1463))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)))] = (((7 <= (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 63)) && ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 63) < 56)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) / 63) * 49)) + (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 63)) - 7))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 1))] = (((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) % 63) < 56)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) % 63)) - 7))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 2))] = (((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) % 63) < 56)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) % 63)) - 7))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 3))] = (((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) % 63) < 56)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) % 63)) - 7))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 4))] = (((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) % 63) < 56)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) % 63)) - 7))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 5))] = (((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) % 63) < 56)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) % 63)) - 7))] : 0.000000e+00f);
    if (((((int)threadIdx.z) * 6) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) / 63)) < 24) {
      if (((((int)threadIdx.z) * 54) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) / 7)) < 216) {
        if ((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) < 1506) {
          if (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) < 372) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 6))] = (((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) % 63) < 56)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) % 63)) - 7))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 6) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) / 63)) < 24) {
      if (((((int)threadIdx.z) * 54) + (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) / 7)) < 215) {
        if ((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) < 1505) {
          if (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) < 371) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 7))] = (((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) % 63) < 56)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) % 63)) - 7))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) / 72) + ((int)threadIdx.z)) < 4) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) / 3)) < 96) {
        if ((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)) < 288) {
          if (((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) < 72) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + (((int)threadIdx.x) * 6)) + 1))];
            }
          }
        }
      }
    }
    if ((((((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) + 1) / 72) + ((int)threadIdx.z)) < 4) {
      if (((((int)threadIdx.z) * 24) + ((((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) + 1) / 3)) < 96) {
        if ((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)) < 287) {
          if (((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) < 71) {
            if (((int)threadIdx.x) < 5) {
              kernel_shared[(((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)) + 1))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + (((int)threadIdx.x) * 6)) + 4))];
            }
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 7))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 14))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 63))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 70))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 77))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 126))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 133))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 140))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 189))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 196))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 203))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 252))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 259))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 266))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 315))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 322))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 329))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 378))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 385))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 392))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 441))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 448))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 455))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 504))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 511))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 518))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 567))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 574))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 581))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 630))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 637))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 644))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 693))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 700))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 707))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 756))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 763))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 770))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 819))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 826))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 833))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 882))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 889))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 896))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 945))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 952))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 959))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1008))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1015))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1022))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1071))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1078))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1085))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1134))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1141))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1148))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1197))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1204))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1211))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1260))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1267))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1274))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1323))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1330))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1337))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1386))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1393))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1400))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1449))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1456))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1463))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    __syncthreads();
    pad_temp_shared[((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)))] = ((((7 <= (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 63)) && ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 63) < 56)) && ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 7) < 6)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) / 63) * 49)) + (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 63)) - 6))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 1))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) % 63) < 56)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) % 7) < 6)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 1) % 63)) - 6))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 2))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) % 63) < 56)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) % 7) < 6)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 2) % 63)) - 6))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 3))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) % 63) < 56)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) % 7) < 6)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 3) % 63)) - 6))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 4))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) % 63) < 56)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) % 7) < 6)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 4) % 63)) - 6))] : 0.000000e+00f);
    pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 5))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) % 63) < 56)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) % 7) < 6)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 5) % 63)) - 6))] : 0.000000e+00f);
    if (((((int)threadIdx.z) * 6) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) / 63)) < 24) {
      if (((((int)threadIdx.z) * 54) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) / 7)) < 216) {
        if ((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) < 1506) {
          if (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) < 372) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 6))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) % 63) < 56)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) % 7) < 6)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 6) % 63)) - 6))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if (((((int)threadIdx.z) * 6) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) / 63)) < 24) {
      if (((((int)threadIdx.z) * 54) + (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) / 7)) < 215) {
        if ((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) < 1505) {
          if (((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) < 371) {
            if (((int)threadIdx.x) < 6) {
              pad_temp_shared[(((((((int)threadIdx.z) * 378) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 8)) + 7))] = ((((7 <= ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) % 63)) && (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) % 63) < 56)) && ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) % 7) < 6)) ? data[((((((rc_outer * 1176) + (((int)threadIdx.z) * 294)) + (((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) / 63) * 49)) + ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 8)) + 7) % 63)) - 6))] : 0.000000e+00f);
            }
          }
        }
      }
    }
    if (((((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) / 72) + ((int)threadIdx.z)) < 4) {
      if (((((int)threadIdx.z) * 24) + (((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) / 3)) < 96) {
        if ((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)) < 288) {
          if (((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) < 72) {
            if (((int)threadIdx.x) < 6) {
              kernel_shared[((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + (((int)threadIdx.x) * 6)) + 2))];
            }
          }
        }
      }
    }
    if ((((((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) + 1) / 72) + ((int)threadIdx.z)) < 4) {
      if (((((int)threadIdx.z) * 24) + ((((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) + 1) / 3)) < 96) {
        if ((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)) < 287) {
          if (((((int)threadIdx.y) * 11) + (((int)threadIdx.x) * 2)) < 71) {
            if (((int)threadIdx.x) < 5) {
              kernel_shared[(((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 11)) + (((int)threadIdx.x) * 2)) + 1))] = kernel[(((((((((int)blockIdx.z) * 6912) + (((int)threadIdx.z) * 1728)) + (rc_outer * 216)) + (((int)threadIdx.y) * 33)) + (((int)threadIdx.x) * 6)) + 5))];
            }
          }
        }
      }
    }
    __syncthreads();
    pad_temp_shared_local[(0)] = pad_temp_shared[(((((int)threadIdx.y) * 7) + ((int)threadIdx.x)))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 7))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 14))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 63))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 70))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 77))];
    kernel_shared_local[(0)] = kernel_shared[((((int)threadIdx.z) * 72))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 1))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 2))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 3))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 4))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 5))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 126))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 133))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 140))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 189))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 196))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 203))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 6))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 7))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 8))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 9))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 10))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 11))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 252))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 259))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 266))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 315))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 322))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 329))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 12))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 13))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 14))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 15))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 16))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 17))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 378))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 385))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 392))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 441))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 448))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 455))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 18))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 19))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 20))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 21))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 22))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 23))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 504))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 511))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 518))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 567))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 574))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 581))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 24))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 25))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 26))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 27))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 28))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 29))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 630))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 637))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 644))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 693))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 700))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 707))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 30))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 31))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 32))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 33))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 34))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 35))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 756))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 763))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 770))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 819))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 826))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 833))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 36))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 37))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 38))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 39))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 40))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 41))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 882))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 889))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 896))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 945))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 952))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 959))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 42))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 43))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 44))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 45))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 46))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 47))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1008))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1015))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1022))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1071))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1078))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1085))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 48))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 49))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 50))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 51))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 52))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 53))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1134))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1141))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1148))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1197))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1204))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1211))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 54))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 55))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 56))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 57))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 58))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 59))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1260))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1267))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1274))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1323))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1330))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1337))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 60))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 61))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 62))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 63))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 64))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 65))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
    pad_temp_shared_local[(0)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1386))];
    pad_temp_shared_local[(1)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1393))];
    pad_temp_shared_local[(2)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1400))];
    pad_temp_shared_local[(3)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1449))];
    pad_temp_shared_local[(4)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1456))];
    pad_temp_shared_local[(5)] = pad_temp_shared[((((((int)threadIdx.y) * 7) + ((int)threadIdx.x)) + 1463))];
    kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 72) + 66))];
    kernel_shared_local[(1)] = kernel_shared[(((((int)threadIdx.z) * 72) + 67))];
    kernel_shared_local[(2)] = kernel_shared[(((((int)threadIdx.z) * 72) + 68))];
    kernel_shared_local[(3)] = kernel_shared[(((((int)threadIdx.z) * 72) + 69))];
    kernel_shared_local[(4)] = kernel_shared[(((((int)threadIdx.z) * 72) + 70))];
    kernel_shared_local[(5)] = kernel_shared[(((((int)threadIdx.z) * 72) + 71))];
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(1)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(2)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(3)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(4)]));
    compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(5)]));
  }
  compute[(((((((int)blockIdx.z) * 196) + (((int)threadIdx.z) * 49)) + (((int)threadIdx.y) * 7)) + ((int)threadIdx.x)))] = compute_local[(0)];
}





float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
    #pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
void pad_input(float * x, float *y){
    #pragma omp parallel for
    for(unsigned int i=0;i<(H + 2)*(W+2)*C;++i){
        y[i] = 0.0f;
    }
    #pragma omp parallel for
    for(unsigned int c=0;c<C;++c){
        for(unsigned int h=0;h<H;++h){
            for(unsigned int w=0;w<W;++w){
                unsigned int h_padded = h + 1;
                unsigned int w_padded = w + 1;
                y[c*(H+2)*(W+2) + h_padded*(W+2) + w_padded] = x[c*(H)*(W) + h*(W) + w];
            }
        }
    }
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float * padded_input = new float[C*(H+2)*(W+2)];
    pad_input(input, padded_input);
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);


    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);

    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);

    dim3 grid(1,1,40);
    dim3 block(7,7,4);

    float * paddedInputDevice;
    chkerr(hipMalloc(&paddedInputDevice, C * (H + 2) * (W + 2) * sizeof(float)));
    chkerr(hipMemcpy(paddedInputDevice, padded_input, C * (H + 2) * (W + 2) * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<endl;
    return 0;
}


