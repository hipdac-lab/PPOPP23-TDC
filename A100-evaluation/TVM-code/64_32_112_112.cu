#include <cudnn.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <malloc.h>
#include <cstdlib>
#include <time.h>
#include <iostream>
#include <sys/types.h>
#include <errno.h>
#include <vector>
#include <fstream>
#include <string>
#include <omp.h>

#define C 64
#define N 32
#define H 112
#define W 112

#define R 3
#define S 3
using namespace std;
#define checkCUDNN(expression)                               \
  {                                                          \
    cudnnStatus_t status = (expression);                     \
    if (status != CUDNN_STATUS_SUCCESS) {                    \
      std::cerr << "Error on line " << __LINE__ << ": "      \
                << cudnnGetErrorString(status) << std::endl; \
      std::exit(EXIT_FAILURE);                               \
    }                                                        \
  }
inline void chkerr(hipError_t code)
{
    if (code != hipSuccess)
    {
        std::cerr << "ERROR!!!:" << hipGetErrorString(code) <<endl;
        exit(-1);
    }
}
class ConvGemm{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvGemm::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvGemm::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvWinogradeNon{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvWinogradeNon::initialize(){
    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvWinogradeNon::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
class ConvFFT{
public:
    float *cpuKernel;
    float alpha = 1.0f;
    float beta = 0.0f;
    cudnnHandle_t convCudnn;
    void* d_workspace{nullptr};
    size_t workspace_bytes{0};
    cudnnTensorDescriptor_t convInputDescriptor;
    cudnnTensorDescriptor_t convOutputDescriptor;
    cudnnFilterDescriptor_t convKernelDescriptor;
    cudnnConvolutionDescriptor_t convDesc;
    float *output;
    float *kernel;
    void initialize();
    float *forward(float *input);
};
void ConvFFT::initialize(){

    hipMalloc(&kernel,sizeof(float)*C*N*9);
    hipMalloc(&this->output,sizeof(float)*N*H*W);
    cudnnCreate(&convCudnn);
    cudnnCreateTensorDescriptor(&convInputDescriptor);
    cudnnSetTensor4dDescriptor(convInputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/C,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnCreateFilterDescriptor(&convKernelDescriptor);
    cudnnSetFilter4dDescriptor(convKernelDescriptor,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*out_channels=*/N,
            /*in_channels=*/C,
            /*kernel_height=*/R,
            /*kernel_width=*/S);
    cudnnCreateConvolutionDescriptor(&convDesc);
    cudnnSetConvolution2dDescriptor(convDesc,
            /*pad_height=*/1,
            /*pad_width=*/1,
            /*vertical_stride=*/1,
            /*horizontal_stride=*/1,
            /*dilation_height=*/1,
            /*dilation_width=*/1,
            /*mode=*/CUDNN_CROSS_CORRELATION,
                                    CUDNN_DATA_FLOAT);
    int batch_size{0}, channels{0}, height{0}, width{0};
    cudnnGetConvolution2dForwardOutputDim(convDesc,
                                          convInputDescriptor,
                                          convKernelDescriptor,
                                          &batch_size,
                                          &channels,
                                          &height,
                                          &width);
    cudnnCreateTensorDescriptor(&convOutputDescriptor);
    cudnnSetTensor4dDescriptor(convOutputDescriptor,
            /*format=*/CUDNN_TENSOR_NCHW,
            /*dataType=*/CUDNN_DATA_FLOAT,
            /*batch_size=*/1,
            /*channels=*/N,
            /*image_height=*/H,
            /*image_width=*/W);
    cudnnGetConvolutionForwardWorkspaceSize(convCudnn,
                                            convInputDescriptor,
                                            convKernelDescriptor,
                                            convDesc,
                                            convOutputDescriptor,
                                            CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                            &workspace_bytes);
    hipMalloc(&d_workspace, workspace_bytes);
    unsigned int kernelSize = R*S*C*N;//kernel
    this->cpuKernel = (float *)malloc(kernelSize*sizeof(float));
    for(int i=0;i<kernelSize;++i){
        this->cpuKernel[i] = 1.0f;
    }
    hipMemcpy(kernel,cpuKernel,R*S*C*N*sizeof(float),hipMemcpyHostToDevice);
    free(cpuKernel);
}
float * ConvFFT::forward(float *input) {
    hipMemset(output, 0, 1*N*H*W*sizeof(float));
    checkCUDNN(cudnnConvolutionForward(convCudnn,
                                       &alpha,
                                       convInputDescriptor,
                                       input,
                                       convKernelDescriptor,
                                       kernel,
                                       convDesc,
                                       CUDNN_CONVOLUTION_FWD_ALGO_FFT,
                                       d_workspace,
                                       workspace_bytes,
                                       &beta,
                                       convOutputDescriptor,
                                       output));
    return output;
}
extern "C" __global__ void default_function_kernel0(float* __restrict__ data, float* __restrict__ kernel, float* __restrict__ compute) {
  float compute_local[16];
  __shared__ float pad_temp_shared[216];
  __shared__ float kernel_shared[144];
  float pad_temp_shared_local[12];
  float kernel_shared_local[12];
  for (int ff_c_init = 0; ff_c_init < 4; ++ff_c_init) {
    for (int xx_c_init = 0; xx_c_init < 2; ++xx_c_init) {
      compute_local[(((ff_c_init * 2) + xx_c_init))] = 0.000000e+00f;
      compute_local[((((ff_c_init * 2) + xx_c_init) + 8))] = 0.000000e+00f;
    }
  }
  for (int rc_outer = 0; rc_outer < 32; ++rc_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 7; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      if (((((((int)threadIdx.y) * 3) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 18)) / 6) + ((int)threadIdx.z)) < 2) {
        if ((((((int)threadIdx.z) * 6) + (((int)threadIdx.y) * 3)) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 18)) < 12) {
          if (((((((int)threadIdx.z) * 108) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 7)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 216) {
            if ((((((int)threadIdx.y) * 54) + (((int)threadIdx.x) * 7)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 108) {
              if (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 54) {
                pad_temp_shared[(((((((int)threadIdx.z) * 108) + (((int)threadIdx.y) * 54)) + (((int)threadIdx.x) * 7)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((1 <= (((((int)blockIdx.y) * 4) + (((int)threadIdx.y) * 3)) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 18))) && ((((((int)blockIdx.y) * 4) + (((int)threadIdx.y) * 3)) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 18)) < 113)) && (1 <= ((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 18)))) && (((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 18)) < 113)) ? data[(((((((((rc_outer * 25088) + (((int)threadIdx.z) * 12544)) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 336)) + ((((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 18) * 112)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 18)) - 113))] : 0.000000e+00f);
              }
            }
          }
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 5; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
      if ((((((int)threadIdx.z) * 4) + (((int)threadIdx.y) * 2)) + (((((int)threadIdx.x) * 5) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 18)) < 8) {
        if ((((((int)threadIdx.z) * 8) + (((int)threadIdx.y) * 4)) + (((((int)threadIdx.x) * 5) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 9)) < 16) {
          if ((((((int)threadIdx.z) * 24) + (((int)threadIdx.y) * 12)) + (((((int)threadIdx.x) * 5) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 3)) < 48) {
            if (((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 5)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 144) {
              if ((((((int)threadIdx.y) * 36) + (((int)threadIdx.x) * 5)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 72) {
                if (((((int)threadIdx.x) * 5) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 36) {
                  kernel_shared[(((((((int)threadIdx.z) * 72) + (((int)threadIdx.y) * 36)) + (((int)threadIdx.x) * 5)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = kernel[(((((((((int)blockIdx.z) * 4608) + (((int)threadIdx.z) * 2304)) + (((int)threadIdx.y) * 1152)) + ((((((int)threadIdx.x) * 5) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 18) * 576)) + (rc_outer * 18)) + (((((int)threadIdx.x) * 5) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) % 18)))];
                }
              }
            }
          }
        }
      }
    }
    __syncthreads();
    for (int rc_inner_outer = 0; rc_inner_outer < 2; ++rc_inner_outer) {
      for (int rx_inner_outer = 0; rx_inner_outer < 3; ++rx_inner_outer) {
        for (int ax2 = 0; ax2 < 3; ++ax2) {
          for (int ax3 = 0; ax3 < 2; ++ax3) {
            pad_temp_shared_local[(((ax2 * 2) + ax3))] = pad_temp_shared[(((((((rc_inner_outer * 108) + (ax2 * 18)) + (((int)threadIdx.y) * 18)) + (((int)threadIdx.x) * 2)) + ax3) + rx_inner_outer))];
            pad_temp_shared_local[((((ax2 * 2) + ax3) + 6))] = pad_temp_shared[((((((((rc_inner_outer * 108) + (ax2 * 18)) + (((int)threadIdx.y) * 18)) + (((int)threadIdx.x) * 2)) + ax3) + rx_inner_outer) + 36))];
          }
        }
        for (int ax0 = 0; ax0 < 4; ++ax0) {
          for (int ax21 = 0; ax21 < 3; ++ax21) {
            kernel_shared_local[(((ax0 * 3) + ax21))] = kernel_shared[((((((((int)threadIdx.z) * 72) + (ax0 * 18)) + (rc_inner_outer * 9)) + (ax21 * 3)) + rx_inner_outer))];
          }
        }
        for (int ry_inner_inner = 0; ry_inner_inner < 3; ++ry_inner_inner) {
          for (int ff_c = 0; ff_c < 4; ++ff_c) {
            for (int xx_c = 0; xx_c < 2; ++xx_c) {
              compute_local[(((ff_c * 2) + xx_c))] = (compute_local[(((ff_c * 2) + xx_c))] + (pad_temp_shared_local[(((ry_inner_inner * 2) + xx_c))] * kernel_shared_local[(((ff_c * 3) + ry_inner_inner))]));
              compute_local[((((ff_c * 2) + xx_c) + 8))] = (compute_local[((((ff_c * 2) + xx_c) + 8))] + (pad_temp_shared_local[((((ry_inner_inner * 2) + xx_c) + 6))] * kernel_shared_local[(((ff_c * 3) + ry_inner_inner))]));
            }
          }
        }
      }
    }
  }
  for (int ff_inner_inner_inner = 0; ff_inner_inner_inner < 4; ++ff_inner_inner_inner) {
    for (int xx_inner_inner_inner = 0; xx_inner_inner_inner < 2; ++xx_inner_inner_inner) {
      compute[(((((((((((int)blockIdx.z) * 100352) + (((int)threadIdx.z) * 50176)) + (ff_inner_inner_inner * 12544)) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner))] = compute_local[(((ff_inner_inner_inner * 2) + xx_inner_inner_inner))];
      compute[((((((((((((int)blockIdx.z) * 100352) + (((int)threadIdx.z) * 50176)) + (ff_inner_inner_inner * 12544)) + (((int)blockIdx.y) * 448)) + (((int)threadIdx.y) * 112)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 224))] = compute_local[((((ff_inner_inner_inner * 2) + xx_inner_inner_inner) + 8))];
    }
  }
}





float check_diff(float *x, float *y, unsigned int size){
    float diff = 0.0f;
    #pragma omp parallel for reduction(+ : diff)
    for(unsigned int i=0;i<size;++i){
        diff += abs(x[i] - y[i]);
    }
    return diff;
}
void pad_input(float * x, float *y){
    #pragma omp parallel for
    for(unsigned int i=0;i<(H + 2)*(W+2)*C;++i){
        y[i] = 0.0f;
    }
    #pragma omp parallel for
    for(unsigned int c=0;c<C;++c){
        for(unsigned int h=0;h<H;++h){
            for(unsigned int w=0;w<W;++w){
                unsigned int h_padded = h + 1;
                unsigned int w_padded = w + 1;
                y[c*(H+2)*(W+2) + h_padded*(W+2) + w_padded] = x[c*(H)*(W) + h*(W) + w];
            }
        }
    }
}
int main(void){
    float *input = new float[C*H*W];
    time_t t;
    srand((unsigned) time(&t));
    for(int i =0;i<C*H*W;++i){
        input[i] = rand() % 10;
    }
    float * padded_input = new float[C*(H+2)*(W+2)];
    pad_input(input, padded_input);
    float *device_input;
    hipMalloc(&device_input,C*H*W*sizeof(float));
    hipMemcpy(device_input,input,C*H*W*sizeof(float),hipMemcpyHostToDevice);
    float *K = new float[C*N*9];
    for(int i=0;i<C*N*9;++i){
        K[i] = 1.0f;
    }

    ConvGemm convGemm;
    convGemm.initialize();
    ConvWinogradeNon convWinogradeNon;
    convWinogradeNon.initialize();
    ConvFFT convFFT;
    convFFT.initialize();

    float *out_cudnn;
    float *out_cudnn_host = new float[N*H*W];
    hipEvent_t event_start;
    hipEvent_t event_stop;
    hipEventCreate(&event_start);
    hipEventCreate(&event_stop);
    out_cudnn = convGemm.forward(device_input);
    hipMemcpy(out_cudnn_host,out_cudnn,N*H*W*sizeof(float),hipMemcpyDeviceToHost);
    out_cudnn = convFFT.forward(device_input);
    out_cudnn = convWinogradeNon.forward(device_input);


    float *device_K;
    float *device_out;
    hipMalloc(&device_out,H*W*N*sizeof(float));
    hipMemset(device_out,0,H*W*N*sizeof(float));
    hipMalloc(&device_K,C*N*9*sizeof(float));
    hipMemcpy(device_K,K,C*N*9*sizeof(float),hipMemcpyHostToDevice);

    hipEventRecord(event_start);
    convGemm.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnGemmTime;
    hipEventElapsedTime(&cudnnGemmTime, event_start, event_stop);

    hipEventRecord(event_start);
    convWinogradeNon.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnWinogradeTimeNon;
    hipEventElapsedTime(&cudnnWinogradeTimeNon, event_start, event_stop);

    hipEventRecord(event_start);
    convFFT.forward(device_input);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float cudnnFFTTime;
    hipEventElapsedTime(&cudnnFFTTime, event_start, event_stop);

    dim3 grid(7,28,4);
    dim3 block(8,2,2);

    float * paddedInputDevice;
    chkerr(hipMalloc(&paddedInputDevice, C * (H + 2) * (W + 2) * sizeof(float)));
    chkerr(hipMemcpy(paddedInputDevice, padded_input, C * (H + 2) * (W + 2) * sizeof(float), hipMemcpyHostToDevice));
    hipEventRecord(event_start);
    default_function_kernel0<<<grid, block>>>(device_input, device_K, device_out);
    hipEventRecord(event_stop);
    hipEventSynchronize(event_stop);
    float time_tdc;
    hipEventElapsedTime(&time_tdc, event_start, event_stop);
    float *out_tdc = new float[N*H*W];
    hipMemcpy(out_tdc,device_out,N*H*W*sizeof(float),hipMemcpyDeviceToHost);

    float difference = check_diff(out_cudnn_host, out_tdc, N*H*W);
    cout<<N<<","<<C<<","<<H<<","<<W<<","<<cudnnFFTTime<<","<<cudnnWinogradeTimeNon<<","<<cudnnGemmTime<<","<<time_tdc<<","<<cudnnFFTTime/time_tdc<<","<<cudnnWinogradeTimeNon/time_tdc<<","<<cudnnGemmTime/time_tdc<<endl;
    return 0;
}


